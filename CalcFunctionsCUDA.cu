#include "hip/hip_runtime.h"
//Copyright (C) 2015, NRC "Kurchatov institute", http://www.nrcki.ru/e/engl.html, Moscow, Russia
//Author: Vladislav Neverov, vs-never@hotmail.com, neverov_vs@nrcki.ru
//
//This file is part of XaNSoNS.
//
//XaNSoNS is free software: you can redistribute it and / or modify
//it under the terms of the GNU General Public License as published by
//the Free Software Foundation, either version 3 of the License, or
//(at your option) any later version.
//
//XaNSoNS is distributed in the hope that it will be useful,
//but WITHOUT ANY WARRANTY; without even the implied warranty of
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//GNU General Public License for more details.
//
//You should have received a copy of the GNU General Public License
//along with this program. If not, see <http://www.gnu.org/licenses/>.

//Contains host and device code for the CUDA version of XaNSoNS

#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "typedefs.h"
#ifdef UseCUDA

#include "config.h"
#include "block.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

//Calculates rotational matrix (from CalcFunctions.cpp)
void calcRotMatrix(vect3d <double> * const RM0, vect3d <double> * const RM1, vect3d <double> * const RM2, const vect3d <double> euler, const unsigned int convention);

//some float4 and float 3 functions (float4 used as float3)
inline __device__ __host__ float dot(float3 a, float3 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
inline __device__ __host__ float dot(float3 a, float4 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
inline __device__ __host__ float dot(float4 a, float3 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
inline __host__ __device__ float3 operator+(float3 a, float3 b){ return make_float3(a.x + b.x, a.y + b.y, a.z + b.z); }
inline __host__ __device__ float3 operator-(float3 a, float3 b){ return make_float3(a.x - b.x, a.y - b.y, a.z - b.z); }
inline __host__ __device__ float3 operator*(float3 a, float b){ return make_float3(a.x * b, a.y * b, a.z * b); }
inline __device__ float length(float3 v){ return sqrtf(dot(v, v)); }

//the following functions are used to calculate 2D diffraction patterns
//all the 2D arrays are flattened

/**
	Resets the 2D scattering intensity array

	@param *I   Intensity array
	@param Nq   Size of the scattering vector magnitude mesh (number of rows in the 2D intensity array)
	@param Nfi  Size of the scattering vector polar angle mesh (number of columns in the 2D intensity array)
*/
__global__ void zeroInt2DKernel(float * const I, const unsigned int Nq, const unsigned int Nfi);

/**
	Resets the 2D scattering amplitude arrays (real and imaginary parts)

	@param *Ar  Real part of the 2D scattering amplitude array
	@param *Ai  Imaginary part of the 2D scattering amplitude array
	@param Nq   Size of the scattering vector magnitude mesh (number of rows in the 2D intensity array)
	@param Nfi  Size of the scattering vector polar angle mesh (number of columns in the 2D intensity array)
*/
__global__ void zeroAmp2DKernel(float * const Ar, float * const Ai, const unsigned int Nq, const unsigned int Nfi);

/**
	Computes the 2D scattering intensity using the scattering amplitude
	
	@param *I   Intensity array
	@param *Ar  Real part of the 2D scattering amplitude array
	@param *Ai  Imaginary part of the 2D scattering amplitude array
	@param Nq   Size of the scattering vector magnitude mesh (number of rows in the 2D amplitude array)
	@param Nfi  Size of the scattering vector polar angle mesh (number of columns in the 2D amplitude array)
*/
__global__ void Sum2DKernel(float * const I, const float * const Ar, const float * const Ai, const unsigned int Nq, const unsigned int Nfi);

/**
	Multiplies the 2D scattering intensity by a normalizing factor

	@param *I   Intensity array
	@param Nq   Size of the scattering vector magnitude mesh (number of rows in the 2D intensity array)
	@param Nfi  Size of the scattering vector polar angle mesh (number of columns in the 2D intensity array)
	@param norm Normalizing factor
*/
__global__ void Norm2DKernel(float * const I, const unsigned int Nq, const unsigned int Nfi, const float norm);

/**
	Computes the polarization factor and multiplies the 2D scattering intensity by this factor

	@param *I     Intensity array
	@param Nq     Size of the scattering vector magnitude mesh (number of rows in the 2D intensity array)
	@param Nfi    Size of the scattering vector polar angle mesh (number of columns in the 2D intensity array)
	@param *q     Scattering vector magnitude array
	@param lambda Wavelength of the source
*/
template <unsigned int BlockSize2D> __global__ void PolarFactor2DKernel(float * const I, const unsigned int Nq, const unsigned int Nfi, const float * const q, const float lambda);

/**
	Computes the real and imaginary parts of the 2D x-ray (source == xray) or neutron (source == neutron) scattering amplitude in the polar coordinates(q, q_fi) of the reciprocal space

	@param source xray or neutron
	@param *Ar    Real part of the 2D scattering amplitude array
	@param *Ai	  Imaginary part of the 2D scattering amplitude array
	@param *q     Scattering vector magnitude array
	@param Nq     Size of the scattering vector magnitude mesh (number of rows in the 2D amplitude array)
	@param Nfi    Size of the scattering vector polar angle mesh (number of columns in the 2D amplitude array)
	@param CS[]   Transposed rotational matrix. Defines the orientation of the nanoparticle in the 3D space.
	@param lambda Wavelength of the source
	@param *ra    Atomic coordinate array
	@param Nfin   Number of atoms to compute for in this kernel call (less or equal to the total number of atoms, cause the kernel is called iteratively in the loop)
	@param *FF    X-ray atomic form-factor array (for one kernel call the computations are done only for the atoms of the same chemical element) (NULL if source is neutron)
	@param SL     Neutron scattering length of the current chemical element (for one kernel call the computations are done only for the atoms of the same chemical element) (0 if source is xray)
*/
template <unsigned int BlockSize2D, unsigned int SizeR> __global__ void calcInt2DKernel(const unsigned int source, float * const Ar, float * const Ai, const float * const q, const unsigned int Nq, const unsigned int Nfi, const float3 CS[], const float lambda, const float4 * const ra, const unsigned int Nfin, const float * const FF, const float SL);

/**
	Organazies the computations of the 2D scattering intensity in the polar coordinates (q,q_fi) of the reciprocal space with CUDA

	@param DeviceNUM  CUDA device number
	@param ***I2D     2D scattering intensity array (host). The memory is allocated inside the function.
	@param **I        1D (averaged over the polar angle) scattering intensity array (host). The memory is allocated inside the function.
	@param *cfg       Parameters of simulation
	@param *NatomEl	  Array containing the total number of atoms of each chemical element (host)
	@param *ra        Atomic coordinate array (device)
	@param *dFF       X-ray atomic form-factor array for all chemical elements (device)
	@param SL         Array of neutron scattering lengths for all chemical elements
	@param *dq        Scattering vector magnitude array (device)
*/
void calcInt2DCuda(const int DeviceNUM, double *** const I2D, double ** const I, const config * const cfg, const unsigned int * const NatomEl, const float4 * const ra, const float * const dFF, const vector<double> SL, const float * const dq);

//the following functions are used to calculate the histogram of interatomic distances

/**
	Resets the histogram array (unsigned long long int)

	@param *rij_hist  Histogram of interatomic distances
	@param N          Size of the array
*/
__global__ void zeroHistKernel(unsigned long long int * const rij_hist, const unsigned int N);

/**
	Computes the total histogram (first Nhist elements) using the partial histograms (for the devices with the CUDA compute capability < 2.0)

	@param *rij_hist   Partial histograms of interatomic distances
	@param Nhistcopies Number of the partial histograms to sum
	@param Nfin        Number of bins to compute for one kernel call
*/
__global__ void sumHistKernel(unsigned long long int * const rij_hist, const unsigned int Nhistcopies, const unsigned int Nfin, const unsigned int Nhist);

/**
	Computes the histogram of interatomic distances

	@param *ri         Pointer to the coordinate of the 1st i-th atom in ra array
	@param *rj         Pointer to the coordinate of the 1st j-th atom in ra array
	@param iMax        Total number of i-th atoms for this kernel call
	@param jMax        Total number of j-th atoms for this kernel call
	@param *rij_hist   Histogram of interatomic distances
	@param bin         Width of the histogram bin
	@param Nhistcopies Number of partial histograms to compute (!=1 for the devices with the CUDA compute capability < 2.0 to reduce the number of atomicAdd() calls)
	@param Nhist       Size of the partial histogram of interatomic distances
	@param Rcut2       Square of the cut-off radius (if cfg->cutoff is true)
	@param add         Addendum to the histogram bin. Equals to 2 if cutoff is false, otherwise equals to 2 if j-th atoms belong to the inner sphere or to 1 if to the outer
	@param diag        True if the j-th atoms and the i-th atoms are the same (diagonal) for this kernel call
*/
template <unsigned int BlockSize2D, bool cutoff> __global__ void calcHistKernel(const float4 * const __restrict__ ri, const float4 *  const __restrict__ rj, const unsigned int iMax, const unsigned int jMax, unsigned long long int *const rij_hist, const float bin, const unsigned int Nhistcopies, const unsigned int Nhist, const float Rcut2, const unsigned long long int add, const bool diag);

/**
	Organazies the computations of the histogram of interatomic distances with CUDA 

	@param DeviceNUM   CUDA device number
	@param **rij_hist  Histogram of interatomic distances (device). The memory is allocated inside the function.
	@param *ra         Atomic coordinate array (device)
	@param *NatomEl    Array containing the total number of atoms of each chemical element (host)
	@param *NatomEl_outer  Array containing the number of atoms of each chemical element including the atoms in the outer sphere (only if cfg.cutoff is True) (host)
	@param *cfg      Parameters of simulation	
*/
void calcHistCuda(const int DeviceNUM, unsigned long long int ** const rij_hist, const float4 * const ra, const unsigned int * const NatomEl, const unsigned int * const NatomEl_outer, const config * const cfg);

//the following functions are used to calculate the powder diffraction pattern using the histogram of interatomic distances

/**
	Resets 1D float array of size N

	@param *A  Array
	@param N   Size of the array	
*/
__global__ void zero1DFloatArrayKernel(float * const A, const unsigned int N);

/**
	Computes the total scattering intensity (first Nq elements) from the partials sums computed by different thread blocks

	@param *I    Scattering intensity array
	@param Nq    Resolution of the total scattering intensity (powder diffraction pattern) 
	@param Nsum  Number of parts to sum (equalt to the total number of thread blocks in the grid)
*/
__global__ void sumIKernel(float * const I, const unsigned int Nq, const unsigned int Nsum);

/**
	Adds the diagonal elements (j==i) of the Debye double sum to the x-ray scattering intensity 

	@param *I    Scattering intensity array
	@param *FF   X-ray atomic form-factor array (for one kernel call the computations are done only for the atoms of the same chemical element)
	@param Nq    Resolution of the total scattering intensity (powder diffraction pattern)
	@param N     Total number of atoms of the chemical element for whcich the computations are done 
*/
__global__ void addIKernelXray(float * const I, const float * const FF, const unsigned int Nq, const unsigned int N);

/**
	Adds the diagonal elements (j==i) of the Debye double sum to the neutron scattering intensity 

	@param *I    Scattering intensity array
	@param Nq    Resolution of the total scattering intensity (powder diffraction pattern)
	@param Add   The value to add to the intensity (the result of multiplying the square of the scattering length 
                 to the total number of atoms of the chemical element for whcich the computations are done) 
*/
__global__ void addIKernelNeutron(float * const I, const unsigned int Nq, const float Add);

/**
	Computes the polarization factor and multiplies scattering intensity by this factor

	@param *I     Scattering intensity array
	@param Nq     Size of the scattering intensity array
	@param *q     Scattering vector magnitude array
	@param lambda Wavelength of the source
*/
__global__ void PolarFactor1DKernel(float * const I, const unsigned int Nq, const float * const q, const float lambda);

/**
	Computes the x-ray (source == xray) or neutron (source == neutron) scattering intensity (powder diffraction pattern) using the histogram of interatomic distances

	@param source          xray or neutron
	@param *I              Scattering intensity array
	@param *FFi            X-ray atomic form factor for the i-th atoms (all the i-th atoms are of the same chemical element for one kernel call) (NULL if source is neutron)
	@param *FFj            X-ray atomic form factor for the j-th atoms (all the j-th atoms are of the same chemical element for one kernel call)(NULL if source is neutron)
	@param SLij            Product of the scattering lenghts of i-th j-th atoms (0 if source is xray)
	@param *q              Scattering vector magnitude array
	@param Nq              Size of the scattering intensity array
	@param **rij_hist      Histogram of interatomic distances (device). The memory is allocated inside the function
	@param iBinSt          Starting index of the histogram bin for this kernel call (the kernel is called iteratively in a loop)
	@param Nhist           Size of the partial histogram of interatomic distances
	@param MaxBinsPerBlock Maximum number of histogram bins used by a single thread block
	@param bin             Width of the histogram bin
	@param Rcut            Cutoff radius in A (if cutoff is true)
	@param damping         cfg->damping
*/
template <unsigned int Size> __global__ void calcIntHistKernel(const unsigned int source, float * const I, const float * const FFi, const float * const FFj, const float SLij, const float *const q, const unsigned int Nq, const unsigned long long int *const rij_hist, const unsigned int iBinSt, const unsigned int Nhist, const unsigned int MaxBinsPerBlock, const float bin, const float Rcut, const bool damping);

/**
    Adds the average density correction to the xray scattering intensity when the cut-off is enabled (cfg.cutoff == true)

	@param *I        Scattering intensity array
	@param *q        Scattering vector magnitude array
	@param Nq        Size of the scattering intensity array
	@param *dFF      X-ray atomic form-factor array for all chemical elements
	@param *NatomEl  Array containing the total number of atoms of each chemical element
	@param Nel       Total number of different chemical elements in the nanoparticle
	@param Ntot      Total number of atoms in the nanoparticle
	@param Rcut      Cutoff radius in A (if cutoff is true)
	@param dens      Average atomic density of the nanoparticle
	@param damping   cfg->damping
*/
__global__ void AddCutoffKernelXray(float * const I, const float * const q, const float * const FF, const unsigned int * const NatomEl, const unsigned int Nel, const unsigned int Ntot, const unsigned int Nq, const float Rcut, const float dens, const bool damping);

/**
    Adds the average density correction to the neutron scattering intensity when the cut-off is enabled (cfg.cutoff == true)

	@param *I        Scattering intensity array
	@param *q        Scattering vector magnitude array
	@param Nq        Size of the scattering intensity array
	@param SLaver    Average neutron scattering length of the nanopaticle
	@param Rcut      Cutoff radius in A (if cutoff is true)
	@param dens      Average atomic density of the nanoparticle
	@param damping   cfg->damping
*/
__global__ void AddCutoffKernelNeutron(float * const I, const float * const q, const float SLaver, const unsigned int Nq, const float Rcut, const float dens, const bool damping);

/**
    Adds the average density correction to the scattering intensity when the cut-off is enabled (cfg.cutoff == true)

	@param GSadd     Grid size for AddCutoffKernel... kernels
	@param *dI       Scattering intensity array (device)
	@param *NatomEl  Array containing the total number of atoms of each chemical element (host)
	@param *cfg      Parameters of simulation
	@param *dFF      X-ray atomic form-factor array for all chemical elements (device)
	@param SL        Array of neutron scattering lengths for all chemical elements
	@param *dq       Scattering vector magnitude array (device)
	@param Ntot      Total number of atoms in the nanoparticle
*/
void AddCutoffCUDA(const unsigned int GSadd, float * const dI, const unsigned int *const NatomEl, const config * const cfg, const float * const dFF, const vector<double> SL, const float * const dq, const unsigned int Ntot);

/**
	Organazies the computations of the scattering intensity (powder diffraction pattern) using the histogram of interatomic distances with CUDA

	@param DeviceNUM CUDA device number
	@param **I       Scattering intensity array (host). The memory is allocated inside the function
	@param *rij_hist Histogram of interatomic distances (device).
	@param *NatomEl  Array containing the total number of atoms of each chemical element (host)
	@param *cfg      Parameters of simulation
	@param *dFF      X-ray atomic form-factor array for all chemical elements (device)
	@param SL        Array of neutron scattering lengths for all chemical elements
	@param *dq       Scattering vector magnitude array (device)
	@param Ntot      Total number of atoms in the nanoparticle
*/
void calcInt1DHistCuda(const int DeviceNUM, double ** const I, const unsigned long long int * const rij_hist, const unsigned int *const NatomEl, const config * const cfg, const float * const dFF, const vector<double> SL, const float * const dq, const unsigned int Ntot);

//the following functions are used to calculate the PDFs

/**
	Computes the partial radial distribution function (RDF)

	@param *dPDF     Partial PDF array
	@param *rij_hist Histogram of interatomic distances (device)
	@param Nhist     Size of the partial histogram of interatomic distances
	@param mult      1 / (Ntot * bin_width)
*/
__global__ void calcPartialRDFkernel(float * const dPDF, const unsigned long long int * const rij_hist, const unsigned int Nhist, const float mult);

/**
	Computes the partial pair distribution function (PDF)

	@param *dPDF     Prtial PDF array
	@param *rij_hist Histogram of interatomic distances (device)
	@param Nhist     Size of the partial histogram of interatomic distances
	@param mult      1 / (4 * PI * rho * Ntot * bin_width)
	@param bin       Width of the histogram bin
*/
__global__ void calcPartialPDFkernel(float * const dPDF, const unsigned long long int * const rij_hist, const unsigned int Nhist, const float mult, const float bin);

/**
	Computes the partial reduced pair distribution function (rPDF)

	@param *dPDF     Partial PDF array.
	@param *rij_hist Histogram of interatomic distances (device)
	@param Nhist     Size of the partial histogram of interatomic distances
	@param mult      1 / (Ntot * bin_width)
	@param submult   4 * PI * rho * NatomEl_i * NatomEl_j / SQR(Ntot)
	@param bin       Width of the histogram bin
*/
__global__ void calcPartialRPDFkernel(float * const dPDF, const unsigned long long int * const rij_hist, const unsigned int Nhist, const float mult, const float submult, const float bin);

/**
	Computes the total PDF using the partial PDFs

	@param *dPDF   Total (first Nhist elements) + partial PDF array. The memory is allocated inside the function.
	@param Nstart  Index of the first element of the partial PDF whcih will be added to the total PDF in this kernel call
	@param Nhist   Size of the partial histogram of interatomic distances
	@param multIJ  FF_i(q0) * FF_j(q0) / <FF> (for x-ray) and SL_i * SL_j / <SL> (for neutron)
*/
__global__ void calcPDFkernel(float * const dPDF, const unsigned int Nstart, const unsigned int Nhist, const float multIJ);

/**
	Depending on the computational scenario organazies the computations of the scattering intensity (powder diffraction pattern) or PDF using the histogram of interatomic distances with CUDA

	@param DeviceNUM       CUDA device number
	@param **I             Scattering intensity array (host). The memory is allocated inside the function.
	@param **PDF           PDF array (host). The memory is allocated inside the function.
	@param *cfg            Parameters of simulation
	@param *NatomEl        Array containing the total number of atoms of each chemical element (host)
	@param *NatomEl_outer  Array containing the number of atoms of each chemical element including the atoms in the outer sphere (only if cfg.cutoff is True) (host)
	@param *ra             Atomic coordinate array (device)
	@param *dFF            X-ray atomic form-factor array for all chemical elements (device)
	@param SL              Array of neutron scattering lengths for all chemical elements
	@param *dq             Scattering vector magnitude array (device)
*/
void calcPDFandDebyeCuda(const int DeviceNUM, double ** const I, double ** const PDF, const config * const cfg, const unsigned int * const NatomEl, const unsigned int * const NatomEl_outer, const float4 * const ra, const float * const dFF, const vector<double> SL, const float * const dq);

//the following functions are used to calculate the powder diffraction pattern using the original Debye equation (without the histogram approximation)

/**
	Computes xray (source == xray) or neutron (source == neutron) scattering intensity (powder diffraction pattern) using the histogram of interatomic distances

	@param source  xray or neutron
	@param *I      Scattering intensity array
	@param *FFi    X-ray atomic form factor for the i-th atoms (all the i-th atoms are of the same chemical element for one kernel call) (NULL if source is neutron)
	@param *FFj    X-ray atomic form factor for the j-th atoms (all the j-th atoms are of the same chemical element for one kernel call) (NULL if source is neutron)
	@param SLij    Product of the scattering lenghts of i-th j-th atoms (0 if source is xray)
	@param *q      Scattering vector magnitude array
	@param Nq      Size of the scattering intensity array
	@param *ri     Pointer to the coordinate of the 1st i-th atom in ra array
	@param *rj     Pointer to the coordinate of the 1st j-th atom in ra array
	@param iMax    Total number of i-th atoms for this kernel call
	@param jMax    Total number of j-th atoms for this kernel call
	@param diag    True if the j-th atoms and the i-th atoms are the same (diagonal) for this kernel call
	@param mult    Multiplier, equal to 2 if cutoff is false, otherwise equals to 2 if j-th atoms belong to the inner sphere or to 1 if to the outer
	@param Rcut2   Square fo the cutoff radius in A^2 (float(SQR(cfg->Rcutoff)), if cutoff is true)
	@param damping cfg->damping
*/
template <unsigned int BlockSize2D, bool cutoff> __global__ void calcIntDebyeKernel(const unsigned int source, float * const I, const float * const FFi, const float * const FFj, const float SLij, const float * const q, const unsigned int Nq, const float4 * const ri, const float4 * const rj, const unsigned int iMax, const unsigned int jMax, const bool diag, const float mult, const float Rcut2, const bool damping);

/**
	Organazies the computations of the scattering intensity (powder diffraction pattern) using the original Debye equation (without the histogram approximation) with CUDA

	@param DeviceNUM CUDA device number
	@param **I             Scattering intensity array (host). The memory is allocated inside the function.
	@param *cfg            Parameters of simulation
	@param *NatomEl        Array containing the total number of atoms of each chemical element (host)
	@param *NatomEl_outer  Array containing the number of atoms of each chemical element including the atoms in the outer sphere (only if cfg.cutoff is True) (host)
	@param *ra             Atomic coordinate array (device)
	@param *dFF            X-ray atomic form-factor array for all chemical elements (device)
	@param SL              Array of neutron scattering lengths for all chemical elements
	@param *dq             Scattering vector magnitude array (device)
*/
void calcIntDebyeCuda(const int DeviceNUM, double ** const I, const config * const cfg, const unsigned int * const NatomEl, const unsigned int * const NatomEl_outer, const float4 * const ra, const float * const dFF, const vector<double> SL, const float * const dq);

//the following functions are used to calculate the partial scattering intensities (for each pair of the structural blocks) using the original Debye equation (without the histogram approximation)

/**
	Computes the partial scattering intensity (*Ipart) from the partials sums (*I) computed by different thread blocks

	@param *I     Scattering intensity array (partials sums as computed by thread blocks)
	@param *Ipart Partial scattering intensity array
	@param Nq     Resolution of the total scattering intensity (powder diffraction pattern)
	@param Nsum   Number of parts to sum (equalt to the total number of thread blocks in the grid)
*/
__global__ void sumIpartialKernel(float * const I, float * const Ipart, const unsigned int Nq, const unsigned int Nsum);

/**
	Computes the total scattering intensity (powder diffraction pattern) using the partial scattering intensity

	@param *I     Partial + total (first Nq elements) scattering intensity array
	@param Nq     Resolution of the total scattering intensity (powder diffraction pattern)
	@param Npart  Number of the partial intensities to sum
*/
__global__ void integrateIpartialKernel(float * const I, const unsigned int Nq, const unsigned int Nparts);

/**
	Organazies the computations of the scattering intensity (powder diffraction pattern + partial intensities) using the original Debye equation (without the histogram approximation) with CUDA

	@param DeviceNUM CUDA device number
	@param **I       Partial + total scattering intensity array (host). The memory is allocated inside the function.
	@param *cfg      Parameters of simulation
	@param *NatomEl  Array containing the total number of atoms of each chemical element (host)
	@param *ra       Atomic coordinate array (device)
	@param *dFF      X-ray atomic form-factor array for all chemical elements (device)
	@param SL        Array of neutron scattering lengths for all chemical elements
	@param *dq       Scattering vector magnitude array (device)
	@param *Block    Array of the structural blocks 
*/
void calcIntPartialDebyeCuda(const int DeviceNUM, double ** const I, const config * const cfg, const unsigned int * const NatomEl, const float4 * const ra, const float * const dFF, const vector <double> SL, const float * const dq, const block * const Block);

//the following functions are used to set the CUDA device, copy/delete the data to/from the device memory

/**
	Queries all CUDA devices. Checks and sets the CUDA device number
	Returns 0 if OK and -1 if no CUDA devices found

	@param *DeviceNUM CUDA device number
*/
int SetDeviceCuda(int * const DeviceNUM);

/**
	Copies the atomic coordinates (ra), scattering vector magnitude (q) and the x-ray atomic form-factors (FF) to the device memory	

	@param *q      Scattering vector magnitude (host)
	@param *cfg    Parameters of simulation
	@param *ra     Atomic coordinates (host)
	@param **dra   Atomic coordinates (device). The memory is allocated inside the function
	@param **dFF   X-ray atomic form-factors (device). The memory is allocated inside the function
	@param **dq    Scattering vector magnitude (device). The memory is allocated inside the function
	@param FF      X-ray atomic form-factors (host)
*/
void dataCopyCUDA(const double *const q, const config * const cfg, const vector < vect3d <double> > * const ra, float4 ** const dra, float ** const dFF, float ** const dq, const vector <double*> FF);

/**
	Deletes the atomic coordinates (ra), scattering vector magnitude (dq) and the x-ray atomic form-factors (dFF) from the device memory

	@param *ra    Atomic coordinates (device)
	@param *dFF   X-ray atomic form-factors (device)
	@param *dq    Scattering vector magnitude (device)
	@param Nel    Total number of different chemical elements in the nanoparticle
*/
void delDataFromDevice(float4 * const ra, float * const dFF, float * const dq, const unsigned int Nel);

/**
	Returns the theoretical peak performance of the CUDA device

	@param deviceProp  Device properties object
	@param show        If True, show the device information on screen
*/
unsigned int GetGFLOPS(const hipDeviceProp_t deviceProp, const bool show);

//Returns the theoretical peak performance of the CUDA device
unsigned int GetGFLOPS(const hipDeviceProp_t deviceProp, const bool show = false){
	const unsigned int cc = deviceProp.major * 10 + deviceProp.minor; //compute capability
	const unsigned int MP = deviceProp.multiProcessorCount; //number of multiprocessors
	const unsigned int clockRate = deviceProp.clockRate / 1000; //GPU clockrate
	unsigned int ALUlanes = 64;	
	switch (cc){
	case 10:
	case 11:
	case 12:
	case 13:
		ALUlanes = 8;
		break;
	case 20:
		ALUlanes = 32;
		break;
	case 21:
		ALUlanes = 48;
		break;
	case 30:
	case 35:
	case 37:
		ALUlanes = 192;
		break;
	case 50:
	case 52:
		ALUlanes = 128;
		break;
	case 60:
		ALUlanes = 64;
		break;
	case 61:
	case 62:
		ALUlanes = 128;
		break;
	case 70:
	case 72:
	case 75:
		ALUlanes = 64;
		break;
	}
	unsigned int GFLOPS = MP * ALUlanes * 2 * clockRate / 1000;
	if (show) {
		cout << "GPU name: " << deviceProp.name << "\n";
		cout << "CUDA compute capability: " << deviceProp.major << "." << deviceProp.minor << "\n";
		cout << "Number of multiprocessors: " << MP << "\n";
		cout << "GPU clock rate: " << clockRate << " MHz" << "\n";
		cout << "Theoretical peak performance: " << GFLOPS << " GFLOPs\n" << endl;
	}
	return GFLOPS;
}

//Resets the 2D scattering intensity array
__global__ void zeroInt2DKernel(float * const I, const unsigned int Nq, const unsigned int Nfi){
	const unsigned int iq = blockDim.y * blockIdx.y + threadIdx.y, ifi = blockDim.x * blockIdx.x + threadIdx.x;
	if ((iq < Nq) && (ifi < Nfi))	I[iq*Nfi + ifi] = 0;
}

//Resets the 2D scattering amplitude arrays (real and imaginary parts)
__global__ void zeroAmp2DKernel(float * const Ar, float * const Ai, const unsigned int Nq, const unsigned int Nfi){
	const unsigned int iq = blockDim.y * blockIdx.y + threadIdx.y, ifi = blockDim.x * blockIdx.x + threadIdx.x;
	if ((iq < Nq) && (ifi < Nfi)){
		Ar[iq*Nfi + ifi] = 0;
		Ai[iq*Nfi + ifi] = 0;
	}
}

//Computes the 2D scattering intensity using the scattering amplitude
__global__ void Sum2DKernel(float * const I,const float * const Ar, const float * const Ai, const unsigned int Nq, const unsigned int Nfi){
	const unsigned int iq = blockDim.y * blockIdx.y + threadIdx.y, ifi = blockDim.x * blockIdx.x + threadIdx.x;
	if ((iq < Nq) && (ifi < Nfi))	I[iq * Nfi + ifi] += SQR(Ar[iq * Nfi + ifi]) + SQR(Ai[iq * Nfi + ifi]);
}

//Multiplies the 2D scattering intensity by a normalizing factor
__global__ void Norm2DKernel(float * const I, const unsigned int Nq, const unsigned int Nfi, const float norm){
	const unsigned int iq = blockDim.y * blockIdx.y + threadIdx.y, ifi = blockDim.x * blockIdx.x + threadIdx.x;
	if ((iq < Nq) && (ifi < Nfi))	I[iq * Nfi + ifi] *= norm;
}

//Computes the polarization factor and multiplies the 2D scattering intensity by this factor
template <unsigned int BlockSize2D> __global__ void PolarFactor2DKernel(float * const I, const unsigned int Nq, const unsigned int Nfi, const float * const q, const float lambda){
	const unsigned int iq = BlockSize2D * blockIdx.y + threadIdx.y, ifi = BlockSize2D * blockIdx.x + threadIdx.x;
	const unsigned int iqCopy = BlockSize2D * blockIdx.y + threadIdx.x;
	__shared__ float factor[BlockSize2D];
	if ((threadIdx.y == 0) && (iqCopy < Nq)) {
		//polarization factor is computed only by the threads of the first warp (half-warp for the devices with CC < 2.0) and stored in the shared memory
		const float sintheta = q[iqCopy] * (lambda * 0.25f / PIf);
		const float cos2theta = 1.f - 2.f * SQR(sintheta);
		factor[threadIdx.x] = 0.5f * (1.f + SQR(cos2theta));
	}
	__syncthreads();
	if ((iq < Nq) && (ifi < Nfi)) I[iq * Nfi + ifi] *= factor[threadIdx.y]; 
}

//Computes polarization factor and multiplies scattering intensity by this factor
__global__ void PolarFactor1DKernel(float * const I, const unsigned int Nq, const float * const q, const float lambda){
	const unsigned int iq = blockIdx.x * blockDim.x + threadIdx.x;
	if (iq < Nq)	{
		const float sintheta = q[iq] * (lambda * 0.25f / PIf);
		const float cos2theta = 1.f - 2.f * SQR(sintheta);
		const float factor = 0.5f * (1.f + SQR(cos2theta));
		I[blockIdx.y * Nq + iq] *= factor;
	}
}

//Computes the real and imaginary parts of the 2D x-ray (source == xray) or neutron (source == neutron) scattering amplitude in the polar coordinates(q, q_fi) of the reciprocal space
template <unsigned int BlockSize2D, unsigned int SizeR> __global__ void calcInt2DKernel(const unsigned int source, float * const Ar, float * const Ai, const float * const q, const unsigned int Nq, const unsigned int Nfi, const float3 CS[], const float lambda, const float4 * const ra, const unsigned int Nfin, const float * const FF, const float SL){
	//to avoid bank conflicts for shared memory operations BlockSize2D should be equal to the size of the warp (or half-warp for the devices with the CC < 2.0)
	//SizeR should be a multiple of BlockSize2D
	const unsigned int iq = BlockSize2D * blockIdx.y + threadIdx.y, ifi = BlockSize2D * blockIdx.x + threadIdx.x; //each thread computes only one element of the 2D amplitude matrix
	const unsigned int iqCopy = BlockSize2D * blockIdx.y + threadIdx.x;//copying of the scattering vector magnitude to the shared memory is performed by the threads of the same warp (half-warp)
	__shared__ float lFF[BlockSize2D]; //cache array for the x-ray  atomic from-factors
	__shared__ float qi[BlockSize2D]; //cache array for the scattering vector magnitude
	__shared__ float4 r[SizeR]; //cache array for the atomic coordinates
	if ((threadIdx.y == 0) && (iqCopy < Nq)) qi[threadIdx.x] = q[iqCopy]; //loading scattering vector magnitude to the shared memory (only threads from the third warp (first half of the second warp) are used)
	if ((source == xray) && (threadIdx.y == 2) && (iqCopy < Nq)) lFF[threadIdx.x] = FF[iqCopy]; //loading x-ray atomic form-factors to the shared memory (only threads from the first warp (half-warp) are used)
	__syncthreads(); //synchronizing after loading to the shared memory
	float cosfi = 0, sinfi = 0;
	float3 qv; //scattering vector	
	if ((iq < Nq) && (ifi < Nfi)){//checking the 2d array margins
		__sincosf(ifi * 2.f * PIf / Nfi, &sinfi, &cosfi); //computing sin(fi), cos(fi)
		const float sintheta = 0.25f * lambda * qi[threadIdx.y] / PIf; //q = 4pi/lambda*sin(theta)
		const float costheta = 1.f - SQR(sintheta); //theta in [0, pi/2];
		qv = make_float3(costheta * cosfi, costheta * sinfi, -sintheta) * qi[threadIdx.y]; //computing the scattering vector
		//instead of pre-multiplying the atomic coordinates by the rotational matrix we are pre-multiplying the scattering vector by the transposed rotational matrix (dot(qv,r) will be the same)
		qv = make_float3(dot(qv, CS[0]), dot(qv, CS[1]), dot(qv, CS[2]));
	}
	float lAr = 0, lAi = 0;
	const unsigned int Niter = Nfin / SizeR + BOOL(Nfin % SizeR);//we don't have enough shared memory to load the array of atomic coordinates as a whole, so we do it with iterations
	for (unsigned int iter = 0; iter < Niter; iter++){
		unsigned int NiterFin = MIN(Nfin - iter * SizeR, SizeR); //checking for the margins of the atomic coordinates array
		if (threadIdx.y < SizeR / BlockSize2D) {
			const unsigned int iAtom = threadIdx.y * BlockSize2D + threadIdx.x; 
			if (iAtom < NiterFin) r[iAtom] = ra[iter * SizeR + iAtom]; //loading the atomic coordinates to the shared memory
		}
		__syncthreads(); //synchronizing after loading to shared memory
		if ((iq < Nq) && (ifi < Nfi)){//checking the 2d array margins
			for (unsigned int iAtom = 0; iAtom < NiterFin; iAtom++){
				__sincosf(dot(qv, r[iAtom]), &sinfi, &cosfi); //cos(dot(qv*r)), sin(dot(qv,r))
				lAr += cosfi; //real part of the amplitute
				lAi += sinfi; //imaginary part of the amplitute
			}
		}
		__syncthreads(); //synchronizing before the next loading starts
	}
	if ((iq < Nq) && (ifi < Nfi)){//checking the 2d array margins
		if (source == xray) {
			Ar[iq * Nfi + ifi] += lFF[threadIdx.y] * lAr; //multiplying the real part of the amplitude by the form-factor and writing the results to the global memory
			Ai[iq * Nfi + ifi] += lFF[threadIdx.y] * lAi; //doing the same for the imaginary part of the amplitude
		}
		else {
			Ar[iq * Nfi + ifi] += SL * lAr;
			Ai[iq * Nfi + ifi] += SL * lAi;
		}
	}	
}

//Organazies the computations of the 2D scattering intensity in the polar coordinates(q, q_fi) of the reciprocal space with CUDA
void calcInt2DCuda(const int DeviceNUM, double *** const I2D, double ** const I, const config * const cfg, const unsigned int * const NatomEl, const float4 * const ra, const float * const dFF, const vector<double> SL, const float * const dq){
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM); //getting device information
	const unsigned int GFLOPS = GetGFLOPS(deviceProp); //theoretical peak GPU performance
	const unsigned int BlockSize2D = BlockSize2Dsmall;
	unsigned int MaxAtomsPerLaunch = 0;
	if (deviceProp.kernelExecTimeoutEnabled){ //killswitch is enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel execution time in seconds
		const double k = 4.e-8; // t = k * MaxAtomsPerLaunch * Nq * Nfi / GFLOPS
		MaxAtomsPerLaunch = (unsigned int)((tmax * GFLOPS) / (k * cfg->q.N * cfg->Nfi)); //maximum number of atoms per kernel launch
	}
	dim3 dimBlock(BlockSize2D, BlockSize2D); //2d thread block size
	dim3 dimGrid(cfg->Nfi / BlockSize2D + BOOL(cfg->Nfi % BlockSize2D), cfg->q.N / BlockSize2D + BOOL(cfg->q.N % BlockSize2D)); //grid size
	//2d scattering intensity should be calculated for the preset orientation of the sample (or averaged over multiple orientations specified by mesh)
	double dalpha = (cfg->Euler.max.x - cfg->Euler.min.x) / cfg->Euler.N.x, dbeta = (cfg->Euler.max.y - cfg->Euler.min.y) / cfg->Euler.N.y, dgamma = (cfg->Euler.max.z - cfg->Euler.min.z) / cfg->Euler.N.z;
	if (cfg->Euler.N.x < 2) dalpha = 0;
	if (cfg->Euler.N.y < 2) dbeta = 0;
	if (cfg->Euler.N.z < 2) dgamma = 0;
	float3 CS[3], *dCS; //three rows of the transposed rotational matrix for the host and the device
	hipMalloc(&dCS, 3 * sizeof(float3)); //allocating the device memory for the transposed rotational matrix
	//allocating memory on the device for amplitude and intensity 2D arrays
	//GPU has linear memory, so we stretch 2D arrays into 1D arrays
	float *dI, *dAr, *dAi;
	const unsigned int Nm = cfg->q.N * cfg->Nfi; //dimension of 2D intensity array		
	hipMalloc(&dAr, Nm * sizeof(float));
	hipMalloc(&dAi, Nm * sizeof(float));
	hipMalloc(&dI, Nm * sizeof(float));
	hipDeviceSynchronize(); //synchronizing before calculating the amplitude
	zeroInt2DKernel << <dimGrid, dimBlock >> >(dI, cfg->q.N, cfg->Nfi); //reseting the 2D intensity matrix
	for (unsigned int ia = 0; ia < cfg->Euler.N.x; ia++){
		const double alpha = cfg->Euler.min.x + (ia + 0.5)*dalpha;
		for (unsigned int ib = 0; ib < cfg->Euler.N.y; ib++){
			const double beta = cfg->Euler.min.y + (ib + 0.5)*dbeta;
			for (unsigned int ig = 0; ig < cfg->Euler.N.z; ig++){
				const double gamma = cfg->Euler.min.z + (ig + 0.5)*dgamma;
				const vect3d <double> euler(alpha, beta, gamma);
				vect3d <double> RM0, RM1, RM2; //three rows of the rotational matrix
				calcRotMatrix(&RM0, &RM1, &RM2, euler, cfg->EulerConvention); //calculating the rotational matrix
				CS[0] = make_float3(float(RM0.x), float(RM1.x), float(RM2.x)); //transposing the rotational matrix
				CS[1] = make_float3(float(RM0.y), float(RM1.y), float(RM2.y));
				CS[2] = make_float3(float(RM0.z), float(RM1.z), float(RM2.z));
				hipMemcpy(dCS, CS, 3 * sizeof(float3), hipMemcpyHostToDevice); //copying transposed rotational matrix from the host memory to the device memory 
				zeroAmp2DKernel << <dimGrid, dimBlock >> >(dAr, dAi, cfg->q.N, cfg->Nfi); //reseting 2D amplitude arrays
				hipDeviceSynchronize(); //synchronizing before calculation starts to ensure that amplitude arrays were successfully set to zero
				unsigned int inp = 0;
				for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++){ //looping over chemical elements (or ions)
					if (MaxAtomsPerLaunch) { //killswitch is enabled so MaxAtomsPerLaunch is set
						for (unsigned int i = 0; i < NatomEl[iEl] / MaxAtomsPerLaunch + BOOL(NatomEl[iEl] % MaxAtomsPerLaunch); i++) { //looping over the iterations
							const unsigned int Nst = inp + i*MaxAtomsPerLaunch; //index for the first atom on the current iteration step
							const unsigned int Nfin = MIN(Nst + MaxAtomsPerLaunch, inp + NatomEl[iEl]) - Nst; //index for the last atom on the current iteration step
							//float time; //time control sequence
							//hipEvent_t start, stop;
							//hipEventCreate(&start);
							//hipEventCreate(&stop);
							//hipEventRecord(start, 0);
							if (cfg->source == xray) calcInt2DKernel <BlockSize2Dsmall, 8 * BlockSize2Dsmall> << <dimGrid, dimBlock >> >(xray, dAr, dAi, dq, cfg->q.N, cfg->Nfi, dCS, float(cfg->lambda), ra + Nst, Nfin, dFF + iEl * cfg->q.N, 0);
							else calcInt2DKernel <BlockSize2Dsmall, 8 * BlockSize2Dsmall> << <dimGrid, dimBlock >> >(neutron, dAr, dAi, dq, cfg->q.N, cfg->Nfi, dCS, float(cfg->lambda), ra + Nst, Nfin, NULL, float(SL[iEl]));
							hipDeviceSynchronize(); //synchronizing to ensure that additive operations does not overlap
							//hipEventRecord(stop, 0);
							//hipEventSynchronize(stop);
							//hipEventElapsedTime(&time, start, stop);
							//cout << "calcInt2DKernel execution time is: " << time << " ms\n" << endl;
						}
					}
					else { //killswitch is disabled so we execute the kernels for the entire ensemble of atoms
						const unsigned int Nst = inp;
						const unsigned int Nfin = NatomEl[iEl];
						if (cfg->source == xray) calcInt2DKernel <BlockSize2Dsmall, 8 * BlockSize2Dsmall> << <dimGrid, dimBlock >> >(xray, dAr, dAi, dq, cfg->q.N, cfg->Nfi, dCS, float(cfg->lambda), ra + Nst, Nfin, dFF + iEl * cfg->q.N, 0);
						else calcInt2DKernel <BlockSize2Dsmall, 8 * BlockSize2Dsmall> << <dimGrid, dimBlock >> >(neutron, dAr, dAi, dq, cfg->q.N, cfg->Nfi, dCS, float(cfg->lambda), ra + Nst, Nfin, NULL, float(SL[iEl]));
						hipDeviceSynchronize(); //synchronizing to ensure that additive operations does not overlap
					}
					inp += NatomEl[iEl];
				}				
				Sum2DKernel << <dimGrid, dimBlock >> >(dI, dAr, dAi, cfg->q.N, cfg->Nfi); //calculating the 2d scattering intensity by the scattering amplitude
			}
		}
	}
	hipFree(dCS);
	hipFree(dAr);
	hipFree(dAi);
	unsigned int Ntot = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Ntot += NatomEl[iEl]; //total number of atoms
	const float norm = 1.f / (Ntot*cfg->Euler.N.x*cfg->Euler.N.y*cfg->Euler.N.z); //normalizing factor
	Norm2DKernel << <dimGrid, dimBlock >> >(dI, cfg->q.N, cfg->Nfi, norm); //normalizing the 2d scattering intensity
	hipDeviceSynchronize(); //synchronizing to ensure that multiplying operations does not overlap
	if (cfg->PolarFactor) PolarFactor2DKernel <BlockSize2Dsmall> << <dimGrid, dimBlock >> >(dI, cfg->q.N, cfg->Nfi, dq, float(cfg->lambda));//multiplying the 2d intensity by polar factor
	float * const hI = new float[Nm]; //host array for 2D intensity
	hipMemcpy(hI, dI, Nm*sizeof(float), hipMemcpyDeviceToHost);  //copying the 2d intensity matrix from the device memory to the host memory 
	hipFree(dI);
	*I = new double[cfg->q.N]; //array for 1d scattering intensity I[q] (I2D[q][fi] averaged over polar angle fi)
	*I2D = new double*[cfg->q.N]; //array for 2d scattering intensity 
	for (unsigned int iq = 0; iq < cfg->q.N; iq++){
		(*I)[iq] = 0;
		(*I2D)[iq] = new double[cfg->Nfi];
		for (unsigned int ifi = 0; ifi < cfg->Nfi; ifi++)	{
			(*I2D)[iq][ifi] = double(hI[iq * cfg->Nfi + ifi]);
			(*I)[iq] += (*I2D)[iq][ifi]; //calculating the 1d intensity (averaging I2D[q][fi] over the polar angle fi)
		}
		(*I)[iq] /= cfg->Nfi;
	}
	delete[] hI;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "2D pattern calculation time: " << time/1000 << " s" << endl;
}

//Resets 1D float array of size N
__global__ void zero1DFloatArrayKernel(float * const A, const unsigned int N){
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i<N) A[i]=0;
}

//Adds the diagonal elements(j == i) of the Debye double sum to the x - ray scattering intensity
__global__ void addIKernelXray(float * const I, const float * const FF, const unsigned int Nq, const unsigned int N) {
	const unsigned int iq = blockIdx.x * blockDim.x + threadIdx.x;
	if (iq < Nq)	{
		const float lFF = FF[iq];
		I[iq] += SQR(lFF) * N;
	}
}

//Adds the diagonal elements(j == i) of the Debye double sum to the neutron scattering intensity
__global__ void addIKernelNeutron(float * const I, const unsigned int Nq, const float Add) {
	const unsigned int iq = blockIdx.x * blockDim.x + threadIdx.x;
	if (iq < Nq)	I[iq] += Add;
}

//Computes the total scattering intensity (first Nq elements) from the partials sums computed by different thread blocks
__global__ void sumIKernel(float * const I, const unsigned int Nq, const unsigned int Nsum){
	const unsigned int iq = blockDim.x * blockIdx.x + threadIdx.x;
	if (iq<Nq) {
		for (unsigned int j = 1; j < Nsum; j++)	I[iq] += I[j * Nq + iq];
	}
}

//Resets the histogram array (unsigned long long int)
__global__ void zeroHistKernel(unsigned long long int * const rij_hist, const unsigned int N){
	const unsigned int i=blockDim.x * blockIdx.x + threadIdx.x;
	if (i<N) rij_hist[i]=0;
}	

//Computes the histogram of interatomic distances
template <unsigned int BlockSize2D, bool cutoff> __global__ void calcHistKernel(const float4 *  const __restrict__ ri, const float4 *  const __restrict__ rj, const unsigned int iMax, const unsigned int jMax, unsigned long long int *const rij_hist, const float bin, const unsigned int Nhistcopies, const unsigned int Nhist, const float Rcut2, const unsigned long long int add, const bool diag){
	if ((diag) && (blockIdx.x < blockIdx.y)) return; //we need to calculate inter-atomic distances only for j > i, so if we are in the diagonal grid, all the subdiagonal blocks (for which j < i for all threads) do nothing and return
	const unsigned int jt = threadIdx.x, it = threadIdx.y;
	const unsigned int j = blockIdx.x * BlockSize2D + jt;
	const unsigned int iCopy = blockIdx.y * BlockSize2D + jt; //jt!!! memory transaction are performed by the threads of the same warp to coalesce them
	const unsigned int i = blockIdx.y * BlockSize2D + it;
	unsigned int copyind = 0;
	if (Nhistcopies > 1) copyind = ((it * BlockSize2D + jt) % Nhistcopies) * Nhist; //some optimization for CC < 2.0. Making multiple copies of the histogram array reduces the number of atomicAdd() operations on the same elements.
	__shared__ float4 ris[BlockSize2D], rjs[BlockSize2D]; //cache arrays for atomic coordinates 
	if ((it == 0) && (j < jMax)) rjs[jt] = rj[j]; //copying atomic coordinates for j-th (column) atoms (only the threads of the first half-warp are used)
	if ((it == 2) && (iCopy < iMax)) ris[jt] = ri[iCopy]; //the same for i-th (row) atoms (only the threads of the first half-warp of the second warp for CC < 2.0 are used)
	__syncthreads(); //sync to ensure that copying is complete
	if ((j < jMax) && (i < iMax) && ((j > i) || (!diag))) {
		const float rij2 = SQR(ris[it].x - rjs[jt].x) + SQR(ris[it].y - rjs[jt].y) + SQR(ris[it].z - rjs[jt].z);//calculate square of distance	
		if (cutoff){	
			if (rij2 < Rcut2) {
				const unsigned int index = (unsigned int)(sqrtf(rij2) / bin); //get the index of histogram bin
				atomicAdd(&rij_hist[copyind + index], add); //add +2 or +1 to histogram bin
			}
		}
		else {
			const unsigned int index = (unsigned int)(sqrtf(rij2) / bin); //get the index of histogram bin
			atomicAdd(&rij_hist[copyind + index], add); //add +1 to histogram bin
		}
	}
}

//Computes the total histogram (first Nhist elements) using the partial histograms (for the devices with the CUDA compute capability < 2.0)
__global__ void sumHistKernel(unsigned long long int * const rij_hist, const unsigned int Nhistcopies, const unsigned int Nfin, const unsigned int Nhist){
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nfin){
		for (unsigned int iCopy = 1; iCopy < Nhistcopies; iCopy++)	rij_hist[i] += rij_hist[Nhist * iCopy + i];
	}
}

//Organazies the computations of the histogram of interatomic distances with CUDA 
void calcHistCuda(const int DeviceNUM, unsigned long long int ** const rij_hist, const float4 * const ra, const unsigned int * const NatomEl, const unsigned int * const NatomEl_outer, const config * const cfg){
	const unsigned int BlockSize = BlockSize1Dsmall, BlockSize2D = BlockSize2Dsmall; //size of the thread blocks (256, 16x16)
	const unsigned int NhistEl = (cfg->Nel * (cfg->Nel + 1)) / 2 * cfg->Nhist;//Number of partial (Element1<-->Element2) histograms
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM); //getting the device properties
	const int cc = deviceProp.major * 10 + deviceProp.minor; //device compute capability
	unsigned int Nhistcopies = 1;
	if (cc<20){//optimization for the devices with CC < 2.0
		//atomic operations work very slow for the devices with Tesla architecture as compared with the modern devices
		//we minimize the number of atomic operations on the same elements by making multiple copies of pair-distribution histograms
		size_t free, total;
		hipMemGetInfo(&free, &total); //checking the amount of the free GPU memory	
		Nhistcopies = MIN(BlockSize,(unsigned int)(0.25 * float(free) / (NhistEl * sizeof(unsigned long long int)))); //set optimal number for histogram copies 
		if (!Nhistcopies) Nhistcopies = 1;
	}
	unsigned int GridSizeExecMax = 2048;
	const unsigned int GFLOPS = GetGFLOPS(deviceProp); //theoretical peak GPU performance
	if (deviceProp.kernelExecTimeoutEnabled)	{//killswitch is enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel time execution in seconds
		const double k = 1.e-6; // t = k * GridSizeExecMax^2 * BlockSize2D^2 / GFLOPS
		GridSizeExecMax = MIN((unsigned int)(sqrt(tmax * GFLOPS / k) / BlockSize2D), GridSizeExecMax);
	}
	//total histogram size is equal to the product of: partial histogram size for one pair of elements (Nhist), number of partial histograms ((Nel*(Nel + 1)) / 2), number of histogram copies (Nhistcopies)
	const unsigned int NhistTotal = NhistEl * Nhistcopies;
	hipError_t err = hipMalloc(rij_hist, NhistTotal * sizeof(unsigned long long int));//trying to allocate large amount of memory, check for errors
	if (err != hipSuccess) cout << "Error in calcHistCuda(), hipMalloc(): " << hipGetErrorString(err) << endl;
	const unsigned int GSzero = MIN(65535, NhistTotal / BlockSize + BOOL(NhistTotal % BlockSize));//Size of the grid for zeroHistKernel (it could not be large than 65535)
	//reseting pair-distribution histogram array
	for (unsigned int iter = 0; iter < NhistTotal / BlockSize + BOOL(NhistTotal % BlockSize); iter += GSzero)	zeroHistKernel << < GSzero, BlockSize >> >(*rij_hist + iter*BlockSize, NhistTotal - iter*BlockSize);
	hipDeviceSynchronize();//synchronizing before the calculation starts
	dim3 blockgrid(BlockSize2D, BlockSize2D);//2D thread block size
	const float4 * * const raEl = new const float4*[cfg->Nel];
	raEl[0] = ra;
	for (unsigned int iEl = 1; iEl < cfg->Nel; iEl++) {
		(cfg->cutoff) ? raEl[iEl] = raEl[iEl - 1] + NatomEl_outer[iEl - 1] : raEl[iEl] = raEl[iEl - 1] + NatomEl[iEl - 1];
	}
	const float bin = float(cfg->hist_bin);
	const float Rcut2 = float(SQR(cfg->Rcutoff));
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {
		unsigned int jElSt = iEl;
		if (cfg->cutoff) jElSt = 0;
		for (unsigned int jEl = jElSt; jEl < cfg->Nel; jEl++) {//each time we move to the next pair of elements (iEl,jEl) we also move to the respective part of histogram (Nstart += Nhist)
			unsigned int jAtomST = 0;
			if ((cfg->cutoff) && (jEl < iEl)) jAtomST = NatomEl[jEl];
			unsigned int Nstart = 0;
			(jEl > iEl) ? Nstart = cfg->Nhist * (cfg->Nel * iEl - (iEl * (iEl + 1)) / 2 + jEl) : Nstart = cfg->Nhist * (cfg->Nel * jEl - (jEl * (jEl + 1)) / 2 + iEl);
			for (unsigned int iAtom = 0; iAtom < NatomEl[iEl]; iAtom += BlockSize2D * GridSizeExecMax){
				const unsigned int GridSizeExecY = MIN((NatomEl[iEl] - iAtom) / BlockSize2D + BOOL((NatomEl[iEl] - iAtom) % BlockSize2D), GridSizeExecMax);//Y-size of the grid on the current step
				const unsigned int iMax = MIN(BlockSize2D * GridSizeExecY, NatomEl[iEl] - iAtom);//index of the last i-th (row) atom
				if (iEl == jEl) jAtomST = iAtom;//loop should exclude subdiagonal grids
				for (unsigned int jAtom = jAtomST; jAtom < NatomEl[jEl]; jAtom += BlockSize2D * GridSizeExecMax){
					const unsigned int GridSizeExecX = MIN((NatomEl[jEl] - jAtom) / BlockSize2D + BOOL((NatomEl[jEl] - jAtom) % BlockSize2D), GridSizeExecMax);//X-size of the grid on the current step
					const unsigned int jMax = MIN(BlockSize2D * GridSizeExecX, NatomEl[jEl] - jAtom);//index of the last j-th (column) atom
					dim3 grid(GridSizeExecX, GridSizeExecY);
					bool diag = false;
					if ((iEl == jEl) && (iAtom == jAtom)) diag = true;//checking if we are on the diagonal grid or not
					/*float time;
					hipEvent_t start, stop;
					hipEventCreate(&start);
					hipEventCreate(&stop);
					hipEventRecord(start, 0);*/
					if (cfg->cutoff) calcHistKernel <BlockSize2Dsmall, true> << <grid, blockgrid >> >(raEl[iEl] + iAtom, raEl[jEl] + jAtom, iMax, jMax, *rij_hist + Nstart, bin, Nhistcopies, NhistEl, Rcut2, 2, diag);
					else calcHistKernel <BlockSize2Dsmall, false> << <grid, blockgrid >> >(raEl[iEl] + iAtom, raEl[jEl] + jAtom, iMax, jMax, *rij_hist + Nstart, bin, Nhistcopies, NhistEl, 0, 2, diag);
					if (deviceProp.kernelExecTimeoutEnabled) hipDeviceSynchronize();//the kernel above uses atomic operation, it's hard to predict the execution time of a single kernel, so sync to avoid the killswitch triggering 
					/*hipEventRecord(stop, 0);
					hipEventSynchronize(stop);
					hipEventElapsedTime(&time, start, stop);
					cout << "calcHistKernel execution time is: " << time << " ms\n" << endl;*/
				}
				if (cfg->cutoff) {
					for (unsigned int jAtom = NatomEl[jEl]; jAtom < NatomEl_outer[jEl]; jAtom += BlockSize2D * GridSizeExecMax){
						unsigned int GridSizeExecX = MIN((NatomEl_outer[jEl] - jAtom) / BlockSize2D + BOOL((NatomEl_outer[jEl] - jAtom) % BlockSize2D), GridSizeExecMax);//X-size of the grid on the current step
						unsigned int jMax = MIN(BlockSize2D * GridSizeExecX, NatomEl_outer[jEl] - jAtom);//index of the last j-th (column) atom
						dim3 grid(GridSizeExecX, GridSizeExecY);
						calcHistKernel <BlockSize2Dsmall, true> << <grid, blockgrid >> >(raEl[iEl] + iAtom, raEl[jEl] + jAtom, iMax, jMax, *rij_hist + Nstart, bin, Nhistcopies, NhistEl, Rcut2, 1, false);
						if (deviceProp.kernelExecTimeoutEnabled) hipDeviceSynchronize();//the kernel above uses atomic operation, it's hard to predict the execution time of a single kernel, so sync to avoid the killswitch triggering 
					}
				}
			}
		}
	}
	hipDeviceSynchronize();//synchronizing to ensure that all calculations ended before histogram copies summation starts
	delete[] raEl;
	if (Nhistcopies>1) {//summing the histogram copies
		const unsigned int GSsum = MIN(65535, NhistEl / BlockSize + BOOL(NhistEl % BlockSize));
		for (unsigned int iter = 0; iter < NhistEl / BlockSize + BOOL(NhistEl % BlockSize); iter += GSsum)	sumHistKernel << <GSsum, BlockSize >> >(*rij_hist + iter * BlockSize, Nhistcopies, NhistEl - iter * BlockSize, NhistEl);
	}
	hipDeviceSynchronize();//synchronizing before the further usage of histogram in other functions
}

//Computes the x-ray (source == xray) or neutron (source == neutron) scattering intensity (powder diffraction pattern) using the histogram of interatomic distances
template <unsigned int Size> __global__ void calcIntHistKernel(const unsigned int source, float * const I, const float * const FFi, const float * const FFj, const float SLij, const float *const q, const unsigned int Nq, const unsigned long long int *const rij_hist, const unsigned int iBinSt, const unsigned int Nhist, const unsigned int MaxBinsPerBlock, const float bin, const float Rcut, const bool damping){
	__shared__ long long int Nrij[Size];//cache array for the histogram
	__shared__ float damp[Size];
	Nrij[threadIdx.x] = 0;
	damp[threadIdx.x] = 1.;
	__syncthreads();
	const unsigned int iBegin = iBinSt + blockIdx.x * MaxBinsPerBlock;//first index for histogram bin to process
	const unsigned int iEnd = MIN(Nhist, iBegin + MaxBinsPerBlock);//last index for histogram bin to process
	if (iEnd < iBegin) return;
	const unsigned int Niter = (iEnd - iBegin) / blockDim.x + BOOL((iEnd - iBegin) % blockDim.x);//number of iterations
	for (unsigned int iter = 0; iter < Niter; iter++){//we don't have enough shared memory to load the histogram array as a whole, so we do it with iterations
		const unsigned int NiterFin = MIN(iEnd - iBegin - iter * blockDim.x, blockDim.x);//maximum number of histogram bins on current iteration step
		if (threadIdx.x < NiterFin) {
			const unsigned int index = iBegin + iter * blockDim.x + threadIdx.x;
			Nrij[threadIdx.x] = rij_hist[index]; //loading the histogram array to shared memory
			if (damping) {
				const float rij = ((float)(index) + 0.5f) * bin;//distance that corresponds to the current histogram bin
				const float x = PIf * rij / Rcut;
				damp[threadIdx.x] = __sinf(x) / x;
			}
		}
		__syncthreads();//synchronizing after loading
		for (unsigned int iterq = 0; iterq < (Nq / blockDim.x) + BOOL(Nq % blockDim.x); iterq++) {//if Nq > blockDim.x there will be threads that compute more than one element of the intensity array
			const unsigned int iq = iterq * blockDim.x + threadIdx.x;//index of the intensity array element
			if (iq < Nq) {//checking for the array margin				
				const float lq = q[iq];//copying the scattering vector magnitude to the local memory
				float lI = 0;
				for (unsigned int i = 0; i < NiterFin; i++) {//looping over the histogram bins
					if (Nrij[i]){
						const float qrij = lq * ((float)(iBegin + iter * blockDim.x + i) + 0.5f) * bin + 0.000001f;//distance that corresponds to the current histogram bin
						lI += Nrij[i] * damp[i] * __sinf(qrij) / qrij;//scattering intensity without form factors
					}
				}
				if (source == xray) I[blockIdx.x * Nq + iq] += lI * FFi[iq] * FFj[iq];//multiplying intensity by form-factors and storing the results in global memory
				else I[blockIdx.x * Nq + iq] += lI * SLij;
			}
		}
		__syncthreads();//synchronizing threads before the next iteration step
	}
}

//Adds the average density correction to the xray scattering intensity when the cut-off is enabled (cfg.cutoff == true)
__global__ void AddCutoffKernelXray(float * const I, const float * const q, const float * const FF, const unsigned int * const NatomEl, const unsigned int Nel, const unsigned int Ntot, const unsigned int Nq, const float Rcut, const float dens, const bool damping){
	const unsigned int iq = blockIdx.x * blockDim.x + threadIdx.x;
	if (iq < Nq) {
		float FFaver = 0;
		for (unsigned int iEl = 0; iEl < Nel; iEl++) FFaver += FF[iEl * Nq + iq] * NatomEl[iEl];
		FFaver /= Ntot;
		const float lq = q[iq];
		if (lq > 0.000001f) {
			const float qrcut = lq * Rcut;
			if (damping) I[iq] += 4.f * PIf * Ntot * dens * SQR(FFaver) * SQR(Rcut) * __sinf(qrcut) / (lq * (SQR(qrcut) - SQR(PIf)));
			else I[iq] += 4.f * PIf * Ntot * dens * SQR(FFaver) * (Rcut * __cosf(qrcut) - __sinf(qrcut) / lq) / SQR(lq);
		}
	}
}

//Adds the average density correction to the neutron scattering intensity when the cut-off is enabled (cfg.cutoff == true)
__global__ void AddCutoffKernelNeutron(float * const I, const float * const q, const float SLaver, const unsigned int Ntot, const unsigned int Nq, const float Rcut, const float dens, const bool damping){
	const unsigned int iq = blockIdx.x * blockDim.x + threadIdx.x;
	if (iq < Nq) {
		const float lq = q[iq];
		if (lq > 0.000001f) {
			const float qrcut = lq * Rcut;
			if (damping) I[iq] += 4.f * PIf * Ntot * dens * SQR(SLaver) * SQR(Rcut) * __sinf(qrcut) / (lq * (SQR(qrcut) - SQR(PIf)));
			else I[iq] += 4.f * PIf * Ntot * dens * SQR(SLaver) * (Rcut * __cosf(qrcut) - __sinf(qrcut) / lq) / SQR(lq);
		}
	}
}

//Adds the average density correction to the scattering intensity when the cut-off is enabled (cfg.cutoff == true)
void AddCutoffCUDA(const unsigned int GSadd, float * const dI, const unsigned int *const NatomEl, const config * const cfg, const float * const dFF, const vector<double> SL, const float * const dq, const unsigned int Ntot) {
	if (cfg->source == xray) {
		unsigned int * dNatomEl = NULL;
		hipMalloc(&dNatomEl, cfg->Nel * sizeof(unsigned int));
		hipMemcpy(dNatomEl, NatomEl, cfg->Nel * sizeof(unsigned int), hipMemcpyHostToDevice);
		AddCutoffKernelXray << <GSadd, BlockSize1Dsmall >> >(dI, dq, dFF, dNatomEl, cfg->Nel, Ntot, cfg->q.N, float(cfg->Rcutoff), float(cfg->p0), cfg->damping);
		hipDeviceSynchronize();
		hipFree(dNatomEl);
	}
	else {
		float SLav = 0;
		for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) SLav += float(SL[iEl]) * NatomEl[iEl];
		SLav /= Ntot;
		AddCutoffKernelNeutron << <GSadd, BlockSize1Dsmall >> >(dI, dq, SLav, Ntot, cfg->q.N, float(cfg->Rcutoff), float(cfg->p0), cfg->damping);
		hipDeviceSynchronize();
	}
}

//Organazies the computations of the scattering intensity (powder diffraction pattern) using the histogram of interatomic distances with CUDA
void calcInt1DHistCuda(const int DeviceNUM, double ** const I, const unsigned long long int * const rij_hist, const unsigned int *const NatomEl, const config * const cfg, const float * const dFF, const vector<double> SL, const float * const dq, const unsigned int Ntot){
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM);//getting device properties
	const int cc = deviceProp.major * 10 + deviceProp.minor;//device compute capability
	unsigned int BlockSize = BlockSize1Dlarge;//setting the size of the thread blocks to 1024 (default)	
	if (cc < 30) BlockSize = BlockSize1Dmedium;//setting the size of the thread blocks to 512 for the devices with CC < 3.0
	const unsigned int GridSize = MIN(256, cfg->Nhist / BlockSize + BOOL(cfg->Nhist % BlockSize));
	const unsigned int GFLOPS = GetGFLOPS(deviceProp);//theoretical peak GPU performance
	unsigned int MaxBinsPerBlock = cfg->Nhist / GridSize + BOOL(cfg->Nhist % GridSize);
	if (deviceProp.kernelExecTimeoutEnabled)	{//killswitch is enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel time execution in seconds
		const double k = 1.5e-5; // t = k * Nq * MaxBinsPerBlock / GFLOPS
		MaxBinsPerBlock = MIN((unsigned int)(tmax * GFLOPS / (k * cfg->q.N)), MaxBinsPerBlock);
	}
	float *dI = NULL;//device array for scattering intensity
	const unsigned int Isize = GridSize * cfg->q.N;//each block writes to it's own copy of scattering intensity array
	hipMalloc(&dI, Isize * sizeof(float));//allocating the device memory for the scattering intensity array
	const unsigned int GSzero = MIN(65535, Isize / BlockSize + BOOL(Isize % BlockSize));//grid size for zero1DFloatArrayKernel
	for (unsigned int iter = 0; iter < Isize / BlockSize + BOOL(Isize % BlockSize); iter += GSzero) zero1DFloatArrayKernel << <GSzero, BlockSize >> >(dI + iter*BlockSize, Isize - iter*BlockSize);//reseting intensity array
	hipDeviceSynchronize();//synchronizing before calculation starts
	const unsigned int GSadd = cfg->q.N / BlockSize1Dsmall + BOOL(cfg->q.N % BlockSize1Dsmall);//grid size for addIKernelXray/addIKernelNeutron
	unsigned int Nstart = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {
		if (cfg->source == xray) addIKernelXray << <GSadd, BlockSize1Dsmall >> > (dI, dFF + iEl * cfg->q.N, cfg->q.N, NatomEl[iEl]);//add contribution form diagonal (i==j) elements in Debye sum
		else addIKernelNeutron << <GSadd, BlockSize1Dsmall >> > (dI, cfg->q.N, float(SQR(SL[iEl]) * NatomEl[iEl]));
		hipDeviceSynchronize();//synchronizing before main calculation starts
		for (unsigned int jEl = iEl; jEl < cfg->Nel; jEl++, Nstart += cfg->Nhist){
			for (unsigned int iBin = 0; iBin < cfg->Nhist; iBin += GridSize * MaxBinsPerBlock) {//iterations to avoid killswitch triggering
				/*float time;
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start, 0);*/
				if (cfg->source == xray) {//Xray
					if (cc >= 30) calcIntHistKernel <BlockSize1Dlarge> << <GridSize, BlockSize >> > (xray, dI, dFF + iEl * cfg->q.N, dFF + jEl * cfg->q.N, 0, dq, cfg->q.N, rij_hist + Nstart, iBin, cfg->Nhist, MaxBinsPerBlock, float(cfg->hist_bin), float(cfg->Rcutoff), cfg->damping);
					else calcIntHistKernel <BlockSize1Dmedium> << <GridSize, BlockSize >> > (xray, dI, dFF + iEl * cfg->q.N, dFF + jEl * cfg->q.N, 0, dq, cfg->q.N, rij_hist + Nstart, iBin, cfg->Nhist, MaxBinsPerBlock, float(cfg->hist_bin), float(cfg->Rcutoff), cfg->damping);
				}
				else {//neutron
					if (cc >= 30) calcIntHistKernel <BlockSize1Dlarge> << <GridSize, BlockSize >> > (neutron, dI, NULL, NULL, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, rij_hist + Nstart, iBin, cfg->Nhist, MaxBinsPerBlock, float(cfg->hist_bin), float(cfg->Rcutoff), cfg->damping);
					else calcIntHistKernel <BlockSize1Dmedium> << <GridSize, BlockSize >> > (neutron, dI, NULL, NULL, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, rij_hist + Nstart, iBin, cfg->Nhist, MaxBinsPerBlock, float(cfg->hist_bin), float(cfg->Rcutoff), cfg->damping);
				}
				/*hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&time, start, stop);
				cout << "calcIntHistKernel execution time is: " << time << " ms\n" << endl;*/
				hipDeviceSynchronize();//synchronizing before the next iteration step
			}
		}
	}
	sumIKernel << <GSadd, BlockSize1Dsmall >> >(dI, cfg->q.N, GridSize);//summing intensity copies
	hipDeviceSynchronize();//synchronizing threads before multiplying the intensity by a polarization factor
	if (cfg->cutoff) AddCutoffCUDA(GSadd, dI, NatomEl, cfg, dFF, SL, dq, Ntot);
	if (cfg->PolarFactor) PolarFactor1DKernel << <GSadd, BlockSize1Dsmall >> >(dI, cfg->q.N, dq, float(cfg->lambda));
	float * const hI = new float[cfg->q.N];
	hipMemcpy(hI, dI, cfg->q.N * sizeof(float), hipMemcpyDeviceToHost);//copying intensity array from the device to the host
	hipFree(dI);//deallocating memory for intensity array
	*I = new double[cfg->q.N];
	for (unsigned int iq = 0; iq < cfg->q.N; iq++) (*I)[iq] = double(hI[iq]) / Ntot;//normalizing
	delete[] hI;
}

//Computes the partial radial distribution function (RDF)
__global__ void calcPartialRDFkernel(float * const dPDF, const unsigned long long int * const rij_hist, const unsigned int Nhist, const float mult) {
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nhist) dPDF[i] = rij_hist[i] * mult;
}

//Computes the partial pair distribution function (PDF)
__global__ void calcPartialPDFkernel(float * const dPDF, const unsigned long long int * const rij_hist, const unsigned int iStart, const unsigned int Nhist, const float mult, const float bin) {
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nhist) {
		const float r = (iStart + i + 0.5f) * bin;
		dPDF[i] = rij_hist[i] * (mult / SQR(r));
	}
}

//Computes the partial reduced pair distribution function(rPDF)
__global__ void calcPartialRPDFkernel(float * const dPDF, const unsigned long long int * const rij_hist, const unsigned int iStart, const unsigned int Nhist, const float mult, const float submult, const float bin) {
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nhist) {
		const float r = (iStart + i + 0.5f) * bin;
		dPDF[i] = rij_hist[i] * (mult / r) - submult * r;
	}
}

//Computes the total PDF using the partial PDFs
__global__ void calcPDFkernel(float * const dPDF, const unsigned int Nstart, const unsigned int Nhist, const float multIJ) {
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nhist) 	dPDF[i] += dPDF[Nstart + i] * multIJ;
}

//Depending on the computational scenario organazies the computations of the scattering intensity (powder diffraction pattern) or PDF using the histogram of interatomic distances with CUDA
void calcPDFandDebyeCuda(const int DeviceNUM, double ** const I, double ** const PDF, const config * const cfg, const unsigned int * const NatomEl, const unsigned int * const NatomEl_outer, const float4 * const ra, const float * const dFF, const vector<double> SL, const float * const dq) {
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	unsigned long long int *rij_hist = NULL;//array for pair-distribution histogram (device only)
	calcHistCuda(DeviceNUM, &rij_hist, ra, NatomEl, NatomEl_outer, cfg);//calculating the histogram
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "Histogram calculation time: " << time / 1000 << " s" << endl;
	unsigned int Ntot = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Ntot += NatomEl[iEl];//calculating the total number of atoms
	if ((cfg->scenario == PDFonly) || (cfg->scenario == DebyePDF)) {//calculating the PDFs
		hipEventRecord(start, 0);
		const unsigned int BlockSize = BlockSize1Dmedium;
		const unsigned int NPDF = (1 + (cfg->Nel * (cfg->Nel + 1)) / 2) * cfg->Nhist;//total PDF array size (full (cfg->Nhist) + partial (cfg->Nhist*(cfg->Nel*(cfg->Nel + 1)) / 2) )
		float *dPDF = NULL;
		hipMalloc(&dPDF, NPDF * sizeof(float));//allocating the device memory for PDF array
		//the size of the histogram array may exceed the maximum number of thread blocks in the grid (65535 for the devices with CC < 3.0) multiplied by the thread block size (512 for devices with CC < 2.0 or 1024 for others)
		//so any operations on histogram array should be performed iteratively
		const unsigned int GSzero = MIN(65535, NPDF / BlockSize + BOOL(NPDF % BlockSize));//grid size for zero1DFloatArrayKernel
		for (unsigned int iter = 0; iter < NPDF; iter += GSzero * BlockSize)	zero1DFloatArrayKernel << <NPDF / BlockSize + BOOL(NPDF % BlockSize), BlockSize >> >(dPDF + iter, NPDF - iter);//reseting the PDF array
		hipDeviceSynchronize();//synchronizing before calculation starts
		const unsigned int GridSizeMax = cfg->Nhist / BlockSize + BOOL(cfg->Nhist % BlockSize);
		const unsigned int GridSize = MIN(65535, GridSizeMax);//grid size for main kernels
		unsigned int Nstart = 0;
		for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {
			for (unsigned int jEl = iEl; jEl < cfg->Nel; jEl++, Nstart += cfg->Nhist){				
				switch (cfg->PDFtype){
					float mult, sub;
					case typeRDF://calculating partial RDFs
						mult = 1.f / (float(cfg->hist_bin) * Ntot);
						for (unsigned int iter = 0; iter < cfg->Nhist; iter += GridSize * BlockSize)	calcPartialRDFkernel << <GridSize, BlockSize >> > (dPDF + iter + cfg->Nhist + Nstart, rij_hist + iter + Nstart, cfg->Nhist - iter, mult);
						break;
					case typePDF://calculating partial PDFs
						mult = 0.25f / (PIf * float(cfg->hist_bin * cfg->p0) * Ntot);
						for (unsigned int iter = 0; iter < cfg->Nhist; iter += GridSize * BlockSize) calcPartialPDFkernel << <GridSize, BlockSize >> > (dPDF + iter + cfg->Nhist + Nstart, rij_hist + iter + Nstart, iter, cfg->Nhist - iter, mult, float(cfg->hist_bin));
						break;
					case typeRPDF://calculating partial rPDFs
						mult = 1.f / (float(cfg->hist_bin) * Ntot);
						(jEl > iEl) ? sub = 8.f * PIf * float(cfg->p0) * float(NatomEl[iEl]) * float(NatomEl[jEl]) / SQR(float(Ntot)) : sub=4.f * PIf * float(cfg->p0) * SQR(float(NatomEl[iEl])) / SQR(float(Ntot));
						for (unsigned int iter = 0; iter < cfg->Nhist; iter += GridSize * BlockSize) calcPartialRPDFkernel << <GridSize, BlockSize >> > (dPDF + iter + cfg->Nhist + Nstart, rij_hist + iter + Nstart, iter, cfg->Nhist - iter, mult, sub, float(cfg->hist_bin));
						break;
				}
			}
		}
		hipDeviceSynchronize();//synchronizing before calculating the full PDF
		Nstart = cfg->Nhist;
		float Faverage2 = 0;
		for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Faverage2 += float(SL[iEl] * NatomEl[iEl]); //calculating the average form-factor
		Faverage2 /= Ntot;
		Faverage2 *= Faverage2;//and squaring it
		for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {//calculating full PDF by summing partial PDFs
			for (unsigned int jEl = iEl; jEl < cfg->Nel; jEl++, Nstart += cfg->Nhist){
				const float multIJ = float(SL[iEl] * SL[jEl]) / Faverage2;
				for (unsigned int iter = 0; iter < cfg->Nhist; iter += GridSize * BlockSize) calcPDFkernel << <GridSize, BlockSize >> > (dPDF + iter, Nstart, cfg->Nhist - iter, multIJ);
				hipDeviceSynchronize();//synchronizing before adding next partial PDF to the full PDF
			}
		}
		unsigned int NPDFh = NPDF;
		if (!cfg->PrintPartialPDF) NPDFh = cfg->Nhist;//if the partial PDFs are not needed, we are not copying them to the host
		float * const hPDF = new float[NPDFh];
		hipMemcpy(hPDF, dPDF, NPDFh * sizeof(float), hipMemcpyDeviceToHost);//copying the PDF from the device to the host
		*PDF = new double[NPDFh];//resulting array of doubles for PDF
		for (unsigned int i = 0; i < NPDFh; i++) (*PDF)[i] = double(hPDF[i]);//converting into double
		delete[] hPDF;
		if (dPDF != NULL) hipFree(dPDF);
		hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
	    hipEventElapsedTime(&time, start, stop);
	    cout << "PDF calculation time: " << time/1000 << " s" << endl;
	}
	if ((cfg->scenario == Debye_hist) || (cfg->scenario == DebyePDF)) {
		hipEventRecord(start, 0);
		calcInt1DHistCuda(DeviceNUM, I, rij_hist, NatomEl, cfg, dFF, SL, dq, Ntot);//calculating the scattering intensity using the pair-distribution histogram
		hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
	    hipEventElapsedTime(&time, start, stop);
	    cout << "1D pattern calculation time: " << time / 1000 << " s" << endl;
	}
	if (rij_hist != NULL) hipFree(rij_hist);//deallocating memory for pair distribution histogram
}

//Computes xray (source == xray) or neutron (source == neutron) scattering intensity (powder diffraction pattern) using the histogram of interatomic distances
template <unsigned int BlockSize2D, bool cutoff> __global__ void calcIntDebyeKernel(const unsigned int source, float * const I, const float * const FFi, const float * const FFj, const float SLij, const float * const q, const unsigned int Nq, const float4 * const ri, const float4 * const rj, const unsigned int iMax, const unsigned int jMax, const bool diag, const float mult, const float Rcut, const bool damping){
	if ((diag) && (blockIdx.x < blockIdx.y)) return; //we need to calculate inter-atomic distances only for j > i, so if we are in the diagonal grid, all the subdiagonal blocks (for which j < i for all threads) do nothing and return
	const unsigned int jt = threadIdx.x, it = threadIdx.y;
	const unsigned int iCopy = blockIdx.y * BlockSize2D + jt; //jt!!! memory transaction are performed by the threads of the same warp to coalesce them
	unsigned int i = blockIdx.y * BlockSize2D + it;
	unsigned int j = blockIdx.x * BlockSize2D + jt;
	__shared__ float4 ris[BlockSize2D], rjs[BlockSize2D]; //cache arrays for the atomic coordinates
	__shared__ float rij[BlockSize2D][BlockSize2D]; //cache array for inter-atomic distances
	__shared__ float damp[BlockSize2D][BlockSize2D]; //cache array for damping coefficients
	rij[it][jt] = -1.f; //reseting inter-atomic distances array
	damp[it][jt] = 1.f;//if damping == false, all damp coefficients are equal to 1
	if (((diag) && (j <= i)) || ((j >= jMax) || (i >= iMax))) damp[it][jt] = 0;//damping coefficients are also used to zero the contribution of subdiagonal elements in the diagonal blocks
	if ((it == 0) && (j < jMax)) rjs[jt] = rj[j]; //copying the atomic coordinates for j-th (column) atoms (only the threads of the first warp (half-warp for CC < 2.0) are used)
	if ((it == 2) && (iCopy < iMax)) ris[jt] = ri[iCopy]; //the same for i-th (row) atoms (only the threads of the third warp (first half-warp of the second warp for CC < 2.0) are used)
	__syncthreads(); //synchronizing threads to ensure that the copying is complete
	//calculating distances
	const float Rcut2 = SQR(Rcut);
	if ((j < jMax) && (i < iMax) && ((j > i) || (!diag))) {
		const float rij2 = SQR(ris[it].x - rjs[jt].x) + SQR(ris[it].y - rjs[jt].y) + SQR(ris[it].z - rjs[jt].z);//calculate square of distance	
		if (cutoff){
			if (rij2 < Rcut2) {
				rij[it][jt] = sqrtf(rij2);
				if (damping) {
					const float x = PIf * rij[it][jt] / Rcut;
					damp[it][jt] = __sinf(x) / x;
				}
			}
		}
		else rij[it][jt] = sqrtf(rij2);
	}
	__syncthreads();//synchronizing threads to ensure that the calculation of the distances is complete
	const unsigned int iEnd = MIN(BlockSize2D, iMax - blockIdx.y * BlockSize2D); //last i-th (row) atom index for the current block
	for (unsigned int iterq = 0; iterq < Nq; iterq += SQR(BlockSize2D)) {//if Nq > SQR(BlockSize2D) there will be threads that compute more than one element of the intensity array
		const unsigned int iq = iterq + it * BlockSize2D + jt;
		if (iq < Nq) {//checking for array margin
			float lI = 0;
			const float lq = q[iq];//copying the scattering vector magnitude to the local memory
			for (i = 0; i < iEnd; i++) {
#pragma unroll 8
				for (j = 0; j < BlockSize2D; j++) {
					if (cutoff) {
						if (rij[i][j] > 0) {
							const float qrij = lq * rij[i][j] + 0.000001f;
							lI += damp[i][j] *__sinf(qrij) / qrij;
						}
					}
					else {
						const float qrij = lq * rij[i][j] + 0.000001f;
						lI += damp[i][j] * __sinf(qrij) / qrij;
					}
				}
			}
			if (source == xray) I[Nq * (gridDim.x * blockIdx.y + blockIdx.x) + iq] += mult * lI * FFi[iq] * FFj[iq]; //multiplying the intensity by form-factors and storing the results in the global memory
			else  I[Nq * (gridDim.x * blockIdx.y + blockIdx.x) + iq] += mult * lI * SLij;
		}
	}
}

//Organazies the computations of the scattering intensity(powder diffraction pattern) using the original Debye equation(without the histogram approximation) with CUDA
void calcIntDebyeCuda(const int DeviceNUM, double ** const I, const config * const cfg, const unsigned int * const NatomEl, const unsigned int * const NatomEl_outer, const float4 * const ra, const float * const dFF, const vector<double> SL, const float * const dq){
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM);//getting the device properties
	size_t free, total;
	hipMemGetInfo(&free, &total);//checking the amount of free GPU memory	
	const unsigned int BlockSize2D = BlockSize2Dsmall;//setting block size to 32x32 (default)
	const unsigned int BlockSize = SQR(BlockSize2D);//total number of threads per block
	const unsigned int GFLOPS = GetGFLOPS(deviceProp); //theoretical peak GPU performance
	unsigned int GridSizeExecMax = MIN(128, (unsigned int)(sqrtf(0.5f * free / (cfg->q.N * sizeof(float)))));//we use two-dimensional grid here, so checking the amount of free memory is really important 
	if (deviceProp.kernelExecTimeoutEnabled)	{//killswitch is enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel time execution in seconds
		const double k = 5.e-8; // t = k * GridSizeExecMax^2 * BlockSize2D^2 * cfg->q.N / GFLOPS
		GridSizeExecMax = MIN((unsigned int)(sqrt(tmax * GFLOPS / (k * cfg->q.N)) / BlockSize2D), GridSizeExecMax);
	}
	float *dI = NULL; //device array for scattering intensity
	const unsigned int Isize = SQR(GridSizeExecMax) * cfg->q.N;//total size of the intensity array	
	hipError_t err=hipMalloc(&dI, Isize * sizeof(float));//allocating memory for the intensity array and checking for errors
	if (err != hipSuccess) cout << "Error in calcIntDebyeCuda(), hipMalloc(dI): " << hipGetErrorString(err) << endl;
	const unsigned int GSzero = MIN(65535, Isize / BlockSize + BOOL(Isize % BlockSize));//grid size for zero1DFloatArrayKernel
	for (unsigned int iter = 0; iter < Isize / BlockSize + BOOL(Isize % BlockSize); iter += GSzero) zero1DFloatArrayKernel << <GSzero, BlockSize >> >(dI + iter*BlockSize, Isize - iter*BlockSize);//reseting the intensity array
	hipDeviceSynchronize();//synchronizing before calculation starts
	dim3 blockgrid(BlockSize2D, BlockSize2D);
	const unsigned int GSadd = cfg->q.N / BlockSize1Dsmall + BOOL(cfg->q.N % BlockSize1Dsmall);//grid size for addIKernelXray/addIKernelNeutron
	const float4 * * const raEl = new const float4*[cfg->Nel];
	raEl[0] = ra;
	for (unsigned int iEl = 1; iEl < cfg->Nel; iEl++) {
		(cfg->cutoff) ? raEl[iEl] = raEl[iEl - 1] + NatomEl_outer[iEl - 1] : raEl[iEl] = raEl[iEl - 1] + NatomEl[iEl - 1];
	}
	const float Rcut = float(cfg->Rcutoff);
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {
		if (cfg->source == xray) addIKernelXray << <GSadd, BlockSize1Dsmall >> > (dI, dFF + iEl * cfg->q.N, cfg->q.N, NatomEl[iEl]);//adding contribution from diagonal (i==j) elements in Debye sum
		else addIKernelNeutron << <GSadd, BlockSize1Dsmall >> > (dI, cfg->q.N, float(SQR(SL[iEl]) * NatomEl[iEl]));
		hipDeviceSynchronize();//synchronizing before main calculation starts
		unsigned int jElSt = iEl;
		if (cfg->cutoff) jElSt = 0;
		for (unsigned int jEl = jElSt; jEl < cfg->Nel; jEl++) {
			unsigned int jAtomST = 0;
			if ((cfg->cutoff) && (jEl < iEl)) jAtomST = NatomEl[jEl];
			for (unsigned int iAtom = 0; iAtom < NatomEl[iEl]; iAtom += BlockSize2D * GridSizeExecMax){
				const unsigned int GridSizeExecY = MIN((NatomEl[iEl] - iAtom) / BlockSize2D + BOOL((NatomEl[iEl] - iAtom) % BlockSize2D), GridSizeExecMax);//Y-size of grid on current step
				const unsigned int iMax = MIN(BlockSize2D * GridSizeExecY, NatomEl[iEl] - iAtom);//last i-th (row) atom in current grid
				if (iEl == jEl) jAtomST = iAtom;
				for (unsigned int jAtom = jAtomST; jAtom < NatomEl[jEl]; jAtom += BlockSize2D * GridSizeExecMax){
					const unsigned int GridSizeExecX = MIN((NatomEl[jEl] - jAtom) / BlockSize2D + BOOL((NatomEl[jEl] - jAtom) % BlockSize2D), GridSizeExecMax);//X-size of grid on current step
					const unsigned int jMax = MIN(BlockSize2D * GridSizeExecX, NatomEl[jEl] - jAtom);//last j-th (column) atom in current grid
					dim3 grid(GridSizeExecX, GridSizeExecY);
					bool diag = false;
					if ((iEl == jEl) && (iAtom == jAtom)) diag = true;//checking if we are on the diagonal grid or not
					/*float time;
					hipEvent_t start, stop;
					hipEventCreate(&start);
					hipEventCreate(&stop);
					hipEventRecord(start, 0);*/
					if (cfg->cutoff) {
						if (cfg->source == xray) calcIntDebyeKernel <BlockSize2Dsmall, true> << <grid, blockgrid >> > (xray, dI, dFF + iEl * cfg->q.N, dFF + jEl * cfg->q.N, 0, dq, cfg->q.N, raEl[iEl] + iAtom, raEl[jEl] + jAtom, iMax, jMax, diag, 2., Rcut, cfg->damping);
						else calcIntDebyeKernel <BlockSize2Dsmall, true> << <grid, blockgrid >> > (neutron, dI, NULL, NULL, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, raEl[iEl] + iAtom, raEl[jEl] + jAtom, iMax, jMax, diag, 2., Rcut, cfg->damping);
					}
					else {
						if (cfg->source == xray) calcIntDebyeKernel <BlockSize2Dsmall, false> << <grid, blockgrid >> > (xray, dI, dFF + iEl * cfg->q.N, dFF + jEl * cfg->q.N, 0, dq, cfg->q.N, raEl[iEl] + iAtom, raEl[jEl] + jAtom, iMax, jMax, diag, 2., 0, false);
						else calcIntDebyeKernel <BlockSize2Dsmall, false> << <grid, blockgrid >> > (neutron, dI, NULL, NULL, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, raEl[iEl] + iAtom, raEl[jEl] + jAtom, iMax, jMax, diag, 2., 0, false);
					}
					hipDeviceSynchronize();//synchronizing before launching next kernel (it will write the data to the same array)
					/*hipEventRecord(stop, 0);
					hipEventSynchronize(stop);
					hipEventElapsedTime(&time, start, stop);
					cout << "calcIntDebyeKernel execution time is: " << time << " ms\n" << endl;*/
				}
				if (cfg->cutoff) {
					for (unsigned int jAtom = NatomEl[jEl]; jAtom < NatomEl_outer[jEl]; jAtom += BlockSize2D * GridSizeExecMax){
						const unsigned int GridSizeExecX = MIN((NatomEl_outer[jEl] - jAtom) / BlockSize2D + BOOL((NatomEl_outer[jEl] - jAtom) % BlockSize2D), GridSizeExecMax);//X-size of grid on current step
						const unsigned int jMax = MIN(BlockSize2D * GridSizeExecX, NatomEl_outer[jEl] - jAtom);//last j-th (column) atom in current grid
						dim3 grid(GridSizeExecX, GridSizeExecY);
						if (cfg->source == xray) calcIntDebyeKernel <BlockSize2Dsmall, true> << <grid, blockgrid >> > (xray, dI, dFF + iEl * cfg->q.N, dFF + jEl * cfg->q.N, 0, dq, cfg->q.N, raEl[iEl] + iAtom, raEl[jEl] + jAtom, iMax, jMax, false, 1., Rcut, cfg->damping);
						else calcIntDebyeKernel <BlockSize2Dsmall, true> << <grid, blockgrid >> > (neutron, dI, NULL, NULL, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, raEl[iEl] + iAtom, raEl[jEl] + jAtom, iMax, jMax, false, 1., Rcut, cfg->damping);
					}
				}
			}
		}
	}
	unsigned int Ntot = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Ntot += NatomEl[iEl]; //calculating total number of atoms
	sumIKernel << <GSadd, BlockSize1Dsmall >> >(dI, cfg->q.N, SQR(GridSizeExecMax));//summing intensity copies
	hipDeviceSynchronize();//synchronizing before multiplying intensity by a polarization factor
	if (cfg->cutoff) AddCutoffCUDA(GSadd, dI, NatomEl, cfg, dFF, SL, dq, Ntot);
	if (cfg->PolarFactor) PolarFactor1DKernel << <GSadd, BlockSize1Dsmall >> >(dI, cfg->q.N, dq, float(cfg->lambda));
	float * const hI = new float[cfg->q.N];
	hipMemcpy(hI, dI, cfg->q.N * sizeof(float), hipMemcpyDeviceToHost);//copying the resulting scattering intensity from the device to the host
	hipFree(dI);//deallocating device memory for intensity array
	*I = new double[cfg->q.N];
	for (unsigned int iq = 0; iq < cfg->q.N; iq++) (*I)[iq] = double(hI[iq]) / Ntot;//normalizing	
	delete[] hI;
	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "1D pattern calculation time: " << time / 1000 << " s" << endl;
}

//Computes the partial scattering intensity (*Ipart) from the partials sums (*I) computed by different thread blocks
__global__ void sumIpartialKernel(float * const I, float * const Ipart, const unsigned int Nq, const unsigned int Nsum){
	const unsigned int iq = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int ipart = blockIdx.y * Nsum * Nq;
	if (iq < Nq) {
		for (unsigned int j = 1; j < Nsum; j++)	I[ipart + iq] += I[ipart + j * Nq + iq];
		Ipart[(blockIdx.y + 1) * Nq + iq] = I[ipart + iq];
	}
}

//Computes the total scattering intensity (powder diffraction pattern) using the partial scattering intensity
__global__ void integrateIpartialKernel(float * const I, const unsigned int Nq, const unsigned int Nparts){
	const unsigned int iq = blockDim.x * blockIdx.x + threadIdx.x;
	if (iq<Nq) {
		I[iq] = 0;
		for (unsigned int ipart = 1; ipart < Nparts + 1; ipart++)	I[iq] += I[ipart * Nq + iq];
	}
}

//Organazies the computations of the scattering intensity (powder diffraction pattern) using the original Debye equation (without the histogram approximation) with CUDA
void calcIntPartialDebyeCuda(const int DeviceNUM, double ** const I, const config * const cfg, const unsigned int * const NatomEl, const float4 * const ra, const float * const dFF, const vector <double> SL, const float * const dq, const block * const Block){
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);	
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM);
	size_t free, total;
	hipMemGetInfo(&free, &total);
	const unsigned int BlockSize2D = BlockSize2Dsmall;
	const unsigned int BlockSize = SQR(BlockSize2D);
	const unsigned int GFLOPS = GetGFLOPS(deviceProp); //theoretical peak GPU performance	
	const unsigned int Nparts = (cfg->Nblocks * (cfg->Nblocks + 1)) / 2;
	unsigned int GridSizeExecMax = MIN(128, (unsigned int)(sqrtf(0.5f * free / (Nparts * cfg->q.N * sizeof(float)))));
	if (deviceProp.kernelExecTimeoutEnabled)	{
		//killswitch enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel time execution in seconds
		const double k = 5.e-8; // t = k * GridSizeExecMax^2 * BlockSize2D^2 / GFLOPS
		GridSizeExecMax = MIN((unsigned int)(sqrt(tmax * GFLOPS / (k * cfg->q.N)) / BlockSize2D), GridSizeExecMax);
	}
	float *dI = NULL;
	const unsigned int IsizeBlock = SQR(GridSizeExecMax) * cfg->q.N;
	const unsigned int Isize = Nparts * IsizeBlock;//each block writes to it's own copy of scattering intensity
	hipError_t err = hipMalloc(&dI, Isize * sizeof(float));
	if (err != hipSuccess) cout << "Error in calcIntPartialDebyeCuda(), hipMalloc(dI): " << hipGetErrorString(err) << endl;
	const unsigned int GSzero = MIN(65535, Isize / BlockSize + BOOL(Isize % BlockSize));
	for (unsigned int iter = 0; iter < Isize / BlockSize + BOOL(Isize % BlockSize); iter += GSzero) zero1DFloatArrayKernel << <GSzero, BlockSize >> >(dI+iter * BlockSize, Isize - iter * BlockSize);	
	dim3 blockgrid(BlockSize2D, BlockSize2D);
	unsigned int * const NatomElBlock = new unsigned int[cfg->Nel * cfg->Nblocks];
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {
		for (unsigned int iB = 0; iB < cfg->Nblocks; iB++){
			NatomElBlock[iEl*cfg->Nblocks + iB] = Block[iB].NatomEl[iEl];
		}
	}	
	const unsigned int GSadd = cfg->q.N / BlockSize1Dsmall + BOOL(cfg->q.N % BlockSize1Dsmall);
	hipDeviceSynchronize();
	unsigned int iAtomST = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iAtomST += NatomEl[iEl], iEl++) {
		for (unsigned int iB = 0; iB < cfg->Nblocks; iB++){
			unsigned int Istart = IsizeBlock * (cfg->Nblocks * iB - (iB * (iB + 1)) / 2 + iB);
			if (cfg->source == xray) addIKernelXray << <GSadd, BlockSize1Dsmall >> > (dI + Istart, dFF + iEl * cfg->q.N, cfg->q.N, NatomElBlock[iEl * cfg->Nblocks + iB]);
			else addIKernelNeutron << <GSadd, BlockSize1Dsmall >> > (dI + Istart, cfg->q.N, float(SQR(SL[iEl]) * NatomElBlock[iEl * cfg->Nblocks + iB]));
		}
		hipDeviceSynchronize();
		unsigned int jAtomST = iAtomST;
		for (unsigned int jEl = iEl; jEl < cfg->Nel; jAtomST += NatomEl[jEl], jEl++) {
			unsigned int iAtomSB = 0;
			for (unsigned int iB = 0; iB < cfg->Nblocks; iAtomSB += NatomElBlock[iEl * cfg->Nblocks + iB], iB++) {
				for (unsigned int iAtom = 0; iAtom < NatomElBlock[iEl * cfg->Nblocks + iB]; iAtom += BlockSize2D*GridSizeExecMax){
					const unsigned int GridSizeExecY = MIN((NatomElBlock[iEl * cfg->Nblocks + iB] - iAtom) / BlockSize2D + BOOL((NatomElBlock[iEl * cfg->Nblocks + iB] - iAtom) % BlockSize2D), GridSizeExecMax);
					const unsigned int iMax = MIN(BlockSize2D * GridSizeExecY, NatomEl[iEl] - iAtom);
					const unsigned int i0 = iAtomST + iAtomSB + iAtom;
					unsigned int jAtomSB = 0;
					for (unsigned int jB = 0; jB < cfg->Nblocks; jAtomSB += NatomElBlock[jEl * cfg->Nblocks + jB], jB++) {
						unsigned int Istart = 0;
						(jB>iB) ? Istart = IsizeBlock * (cfg->Nblocks * iB - (iB * (iB + 1)) / 2 + jB) : Istart = IsizeBlock * (cfg->Nblocks * jB - (jB * (jB + 1)) / 2 + iB);
						for (unsigned int jAtom = 0; jAtom < NatomElBlock[jEl * cfg->Nblocks + jB]; jAtom += BlockSize2D * GridSizeExecMax){
							const unsigned int j0 = jAtomST + jAtomSB + jAtom;
							if (j0 >= i0) {
								const unsigned int GridSizeExecX = MIN((NatomElBlock[jEl * cfg->Nblocks + jB] - jAtom) / BlockSize2D + BOOL((NatomElBlock[jEl * cfg->Nblocks + jB] - jAtom) % BlockSize2D), GridSizeExecMax);
								const unsigned int jMax = MIN(BlockSize2D * GridSizeExecX, NatomElBlock[jEl * cfg->Nblocks + jB] - jAtom);
								dim3 grid(GridSizeExecX, GridSizeExecY);
								bool diag = false;
								if (i0 == j0) diag = true;
								if (cfg->source == xray) calcIntDebyeKernel <BlockSize2Dsmall, false> << <grid, blockgrid >> > (xray, dI + Istart, dFF + iEl * cfg->q.N, dFF + jEl * cfg->q.N, 0, dq, cfg->q.N, ra + i0, ra + j0, iMax, jMax, diag, 2., 0, false);
								else calcIntDebyeKernel <BlockSize2Dsmall, false> << <grid, blockgrid >> > (neutron, dI + Istart, NULL, NULL, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, ra + i0, ra + j0, iMax, jMax, diag, 2., 0, false);
								hipDeviceSynchronize();
							}
						}
					}					
				}
			}			
		}
	}
	delete[] NatomElBlock;
	const unsigned int IpartialSize = (Nparts + 1) * cfg->q.N;
	float *dIpart = NULL;
	hipMalloc(&dIpart, IpartialSize*sizeof(float));
	dim3 gridAdd(GSadd, Nparts);
	sumIpartialKernel << <gridAdd, BlockSize1Dsmall >> >(dI, dIpart, cfg->q.N, SQR(GridSizeExecMax));
	hipDeviceSynchronize();
	hipFree(dI);
	integrateIpartialKernel << <GSadd, BlockSize1Dsmall >> > (dIpart, cfg->q.N, Nparts);
	hipDeviceSynchronize();
	dim3 gridPolar(GSadd, Nparts + 1);
	if (cfg->PolarFactor) PolarFactor1DKernel << <gridPolar, BlockSize1Dsmall >> >(dIpart, cfg->q.N, dq, float(cfg->lambda));
	float * const hI = new float[IpartialSize];
	hipMemcpy(hI, dIpart, IpartialSize * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dIpart);
	*I = new double[IpartialSize];
	unsigned int Ntot = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Ntot += NatomEl[iEl];
	for (unsigned int i = 0; i < IpartialSize; i++) (*I)[i] = double(hI[i]) / Ntot;	
	delete[] hI;
	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "1D pattern calculation time: " << time / 1000 << " s" << endl;
}

//Queries all CUDA devices. Checks and sets the CUDA device number
//Returns 0 if OK and - 1 if no CUDA devices found
int SetDeviceCuda(int * const DeviceNUM){
	int nDevices;
	hipGetDeviceCount(&nDevices);
	if (!nDevices) {
		cout << "Error: No CUDA devices found." << endl;
		return -1;
	}
	if (*DeviceNUM > -1){
		if (*DeviceNUM < nDevices){
			hipSetDevice(*DeviceNUM);
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, *DeviceNUM);
			cout << "Selected CUDA device:" << endl;
			GetGFLOPS(deviceProp, true);
			return 0;
		}
		cout << "Error: Unable to set CUDA device " << *DeviceNUM << ". The total number of CUDA devices is " << nDevices << ".\n";
		cout << "Will use the fastest CUDA device." << endl;
	}
	cout << "The following CUDA devices are found.\n";
	hipDeviceProp_t deviceProp;
	unsigned int GFOLPS=0, MaxGFOLPS=0;
	for (int i = 0; i < nDevices; i++) {
		hipGetDeviceProperties(&deviceProp,i);
		cout << "Device " << i << ":" << endl;
		GFOLPS = GetGFLOPS(deviceProp, true);
		if (GFOLPS > MaxGFOLPS) {
			MaxGFOLPS = GFOLPS;
			*DeviceNUM = i;
		}
	}
	cout << "Will use CUDA device " << *DeviceNUM << "." << endl;
	hipSetDevice(*DeviceNUM);
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	return 0;
}

//Copies the atomic coordinates (ra), scattering vector magnitude (q) and the x-ray atomic form-factors (FF) to the device memory	
void dataCopyCUDA(const double *const q, const config * const cfg, const vector < vect3d <double> > * const ra, float4 ** const dra, float ** const dFF, float ** const dq, const vector <double*> FF){
	//copying the main data to the device memory
	if (cfg->scenario != PDFonly) {
		float * const qfloat = new float[cfg->q.N]; // temporary float array for the scattering vector magnitude
		for (unsigned int iq = 0; iq < cfg->q.N; iq++) qfloat[iq] = (float)q[iq];//converting scattering vector magnitude from double to float
		hipMalloc(dq, cfg->q.N * sizeof(float));//allocating memory for the scattering vector magnitude array
		hipMemcpy(*dq, qfloat, cfg->q.N * sizeof(float), hipMemcpyHostToDevice);//copying scattering vector magnitude array from the host to the device
		delete[] qfloat;//deleting temporary array
		if (cfg->source == xray) {
			hipMalloc(dFF, cfg->q.N * cfg->Nel * sizeof(float));//allocating device memory for the atomic form-factors
			float * const FFfloat = new float[cfg->q.N * cfg->Nel];
			for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++){
				for (unsigned int iq = 0; iq < cfg->q.N; iq++) FFfloat[iEl * cfg->q.N + iq] = float(FF[iEl][iq]);//converting form-factors from double to float				
			}
			hipMemcpy(*dFF, FFfloat, cfg->Nel * cfg->q.N * sizeof(float), hipMemcpyHostToDevice);//copying form-factors from the host to the device
			delete[] FFfloat;//deleting temporary array
		}
	}
	unsigned int Nat = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Nat += (unsigned int)ra[iEl].size();
	hipMalloc(dra, Nat * sizeof(float4));//allocating device memory for the atomic coordinates array
	float4 * const hra = new float4[Nat]; //temporary host array for atomic coordinates
	unsigned int iAtom = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++){
		for (vector<vect3d <double> >::const_iterator ri = ra[iEl].begin(); ri != ra[iEl].end(); ri++, iAtom++){
			hra[iAtom] = make_float4((float)ri->x, (float)ri->y, (float)ri->z, 0);//converting atomic coordinates from vect3d <double> to float4
		}
	}	
	hipMemcpy(*dra, hra, Nat * sizeof(float4), hipMemcpyHostToDevice);//copying atomic coordinates from the host to the device
	delete[] hra;//deleting temporary array
}

//Deletes the atomic coordinates (ra), scattering vector magnitude (dq) and the x-ray atomic form-factors (dFF) from the device memory
void delDataFromDevice(float4 * const ra, float * const dFF,float * const dq, const unsigned int Nel){
	hipFree(ra);//deallocating device memory for the atomic coordinates array
	if (dq != NULL) hipFree(dq);//deallocating memory for the scattering vector magnitude array
	if (dFF != NULL) hipFree(dFF);//deallocating device memory for the atomic form-factors
	hipDeviceReset();//NVIDIA Profiler works improperly without this
}
#endif
